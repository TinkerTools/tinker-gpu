#include "hip/hip_runtime.h"
#include "add.h"
#include "box.h"
#include "e_mpole.h"
#include "launch.cuh"
#include "md.h"
#include "pme.h"
#include "seq_pme.h"


TINKER_NAMESPACE_BEGIN
static constexpr int PME_BLOCKDIM = 64;


enum
{
   PCHG_GRID = 1,
   MPOLE_GRID,
   UIND_GRID,
   UIND_GRID_FPHI2,
   DISP_GRID
};


template <int WHAT, int bsorder>
__global__
void grid_tmpl_cu(const real* restrict x, const real* restrict y,
                  const real* restrict z, int n, int nfft1, int nfft2,
                  int nfft3, const real* restrict ptr1,
                  const real* restrict ptr2, real* restrict qgrid,
                  real3 recip_a, real3 recip_b, real3 recip_c)
{
   real thetai1[4 * MAX_BSORDER];
   real thetai2[4 * MAX_BSORDER];
   real thetai3[4 * MAX_BSORDER];
   __shared__ real sharedarray[MAX_BSORDER * MAX_BSORDER * PME_BLOCKDIM];
   real* restrict array = &sharedarray[MAX_BSORDER * MAX_BSORDER * threadIdx.x];


   MAYBE_UNUSED const real(*fmp)[10] = (real(*)[10])ptr1;
   MAYBE_UNUSED const real(*fuind)[3] = (real(*)[3])ptr1;
   MAYBE_UNUSED const real(*fuinp)[3] = (real(*)[3])ptr2;


   for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n;
        i += blockDim.x * gridDim.x) {
      real xi = x[i];
      real yi = y[i];
      real zi = z[i];


      real w1 = xi * recip_a.x + yi * recip_a.y + zi * recip_a.z;
      w1 = w1 + 0.5f - REAL_FLOOR(w1 + 0.5f);
      real fr1 = nfft1 * w1;
      int igrid1 = REAL_FLOOR(fr1);
      w1 = fr1 - igrid1;


      real w2 = xi * recip_b.x + yi * recip_b.y + zi * recip_b.z;
      w2 = w2 + 0.5f - REAL_FLOOR(w2 + 0.5f);
      real fr2 = nfft2 * w2;
      int igrid2 = REAL_FLOOR(fr2);
      w2 = fr2 - igrid2;


      real w3 = xi * recip_c.x + yi * recip_c.y + zi * recip_c.z;
      w3 = w3 + 0.5f - REAL_FLOOR(w3 + 0.5f);
      real fr3 = nfft3 * w3;
      int igrid3 = REAL_FLOOR(fr3);
      w3 = fr3 - igrid3;


      igrid1 = igrid1 - bsorder + 1;
      igrid2 = igrid2 - bsorder + 1;
      igrid3 = igrid3 - bsorder + 1;
      igrid1 += (igrid1 < 0 ? nfft1 : 0);
      igrid2 += (igrid2 < 0 ? nfft2 : 0);
      igrid3 += (igrid3 < 0 ? nfft3 : 0);


      if CONSTEXPR (WHAT == MPOLE_GRID) {
         bsplgen<3, bsorder>(w1, thetai1, array);
         bsplgen<3, bsorder>(w2, thetai2, array);
         bsplgen<3, bsorder>(w3, thetai3, array);
      }


      if CONSTEXPR (WHAT == UIND_GRID) {
         bsplgen<2, bsorder>(w1, thetai1, array);
         bsplgen<2, bsorder>(w2, thetai2, array);
         bsplgen<2, bsorder>(w3, thetai3, array);
      }


      if CONSTEXPR (WHAT == MPOLE_GRID) {
         real fmpi0 = fmp[i][mpl_pme_0];
         real fmpix = fmp[i][mpl_pme_x];
         real fmpiy = fmp[i][mpl_pme_y];
         real fmpiz = fmp[i][mpl_pme_z];
         real fmpixx = fmp[i][mpl_pme_xx];
         real fmpiyy = fmp[i][mpl_pme_yy];
         real fmpizz = fmp[i][mpl_pme_zz];
         real fmpixy = fmp[i][mpl_pme_xy];
         real fmpixz = fmp[i][mpl_pme_xz];
         real fmpiyz = fmp[i][mpl_pme_yz];
         for (int iz = 0; iz < bsorder; ++iz) {
            int zbase = igrid3 + iz;
            zbase -= (zbase >= nfft3 ? nfft3 : 0);
            zbase *= (nfft1 * nfft2);
            real v0 = thetai3[4 * iz];
            real v1 = thetai3[4 * iz + 1];
            real v2 = thetai3[4 * iz + 2];
            for (int iy = 0; iy < bsorder; ++iy) {
               int ybase = igrid2 + iy;
               ybase -= (ybase >= nfft2 ? nfft2 : 0);
               ybase *= nfft1;
               real u0 = thetai2[4 * iy];
               real u1 = thetai2[4 * iy + 1];
               real u2 = thetai2[4 * iy + 2];
               // fmp: 0, x, y, z, xx, yy, zz, xy, xz, yz
               //      1, 2, 3, 4,  5,  6,  7,  8,  9, 10
               real term0 = fmpi0 * u0 * v0 + fmpiy * u1 * v0 +
                  fmpiz * u0 * v1 + fmpiyy * u2 * v0 + fmpizz * u0 * v2 +
                  fmpiyz * u1 * v1;
               real term1 =
                  fmpix * u0 * v0 + fmpixy * u1 * v0 + fmpixz * u0 * v1;
               real term2 = fmpixx * u0 * v0;
               for (int ix = 0; ix < bsorder; ++ix) {
                  int xbase = igrid1 + ix;
                  xbase -= (xbase >= nfft1 ? nfft1 : 0);
                  int index = xbase + ybase + zbase;
                  real t0 = thetai1[4 * ix];
                  real t1 = thetai1[4 * ix + 1];
                  real t2 = thetai1[4 * ix + 2];
                  atomic_add(term0 * t0 + term1 * t1 + term2 * t2, qgrid,
                             2 * index);
               }
            } // end for (int iy)
         }
      } // end if (WHAT == MPOLE_GRID)


      if CONSTEXPR (WHAT == UIND_GRID) {
         real fuindi0 = fuind[i][0];
         real fuindi1 = fuind[i][1];
         real fuindi2 = fuind[i][2];
         real fuinpi0 = fuinp[i][0];
         real fuinpi1 = fuinp[i][1];
         real fuinpi2 = fuinp[i][2];
         for (int iz = 0; iz < bsorder; ++iz) {
            int zbase = igrid3 + iz;
            zbase -= (zbase >= nfft3 ? nfft3 : 0);
            zbase *= (nfft1 * nfft2);
            real v0 = thetai3[4 * iz];
            real v1 = thetai3[4 * iz + 1];
            for (int iy = 0; iy < bsorder; ++iy) {
               int ybase = igrid2 + iy;
               ybase -= (ybase >= nfft2 ? nfft2 : 0);
               ybase *= nfft1;
               real u0 = thetai2[4 * iy];
               real u1 = thetai2[4 * iy + 1];
               real term01 = fuindi1 * u1 * v0 + fuindi2 * u0 * v1;
               real term11 = fuindi0 * u0 * v0;
               real term02 = fuinpi1 * u1 * v0 + fuinpi2 * u0 * v1;
               real term12 = fuinpi0 * u0 * v0;
               for (int ix = 0; ix < bsorder; ++ix) {
                  int xbase = igrid1 + ix;
                  xbase -= (xbase >= nfft1 ? nfft1 : 0);
                  int index = xbase + ybase + zbase;
                  real t0 = thetai1[4 * ix];
                  real t1 = thetai1[4 * ix + 1];
                  atomic_add(term01 * t0 + term11 * t1, qgrid, 2 * index);
                  atomic_add(term02 * t0 + term12 * t1, qgrid, 2 * index + 1);
               }
            } // end for (int iy)
         }
      } // end if (WHAT == UIND_GRID)
   }
}


void grid_mpole_cu(PMEUnit pme_u, real (*fmp)[10])
{
   auto& st = *pme_u;
   int n1 = st.nfft1;
   int n2 = st.nfft2;
   int n3 = st.nfft3;
   int nt = n1 * n2 * n3;


   device_array::zero_async(2 * nt, st.qgrid);
   auto ker = grid_tmpl_cu<MPOLE_GRID, 5>;
   launch_k2s(nonblk, PME_BLOCKDIM, n, ker, x, y, z, n, n1, n2, n3,
              (const real*)fmp, nullptr, st.qgrid, recipa, recipb, recipc);
}


void grid_uind_cu(PMEUnit pme_u, real (*fuind)[3], real (*fuinp)[3])
{
   auto& st = *pme_u;
   int n1 = st.nfft1;
   int n2 = st.nfft2;
   int n3 = st.nfft3;
   int nt = n1 * n2 * n3;


   device_array::zero_async(2 * nt, st.qgrid);
   auto ker = grid_tmpl_cu<UIND_GRID, 5>;
   launch_k2s(nonblk, PME_BLOCKDIM, n, ker, x, y, z, n, n1, n2, n3,
              (const real*)fuind, (const real*)fuinp, st.qgrid, recipa, recipb,
              recipc);
}


template <int WHAT, int bsorder>
__global__
void fphi_tmpl_cu(int n, int nfft1, int nfft2, int nfft3,
                  const real* restrict x, const real* restrict y,
                  const real* restrict z, real* restrict opt1,
                  real* restrict opt2, real* restrict opt3,
                  const real* restrict qgrid, real3 recip_a, real3 recip_b,
                  real3 recip_c)
{
   real thetai1[4 * MAX_BSORDER];
   real thetai2[4 * MAX_BSORDER];
   real thetai3[4 * MAX_BSORDER];
   real array[MAX_BSORDER * MAX_BSORDER];


   MAYBE_UNUSED real(*restrict fphi)[20] = (real(*)[20])opt1;
   MAYBE_UNUSED real(*restrict fdip_phi1)[10] = (real(*)[10])opt1;
   MAYBE_UNUSED real(*restrict fdip_phi2)[10] = (real(*)[10])opt2;
   MAYBE_UNUSED real(*restrict fdip_sum_phi)[20] = (real(*)[20])opt3;


   for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n;
        i += blockDim.x * gridDim.x) {
      real xi = x[i];
      real yi = y[i];
      real zi = z[i];


      real w1 = xi * recip_a.x + yi * recip_a.y + zi * recip_a.z;
      w1 = w1 + 0.5f - REAL_FLOOR(w1 + 0.5f);
      real fr1 = nfft1 * w1;
      int igrid1 = REAL_FLOOR(fr1);
      w1 = fr1 - igrid1;


      real w2 = xi * xi * recip_b.x + yi * recip_b.y + zi * recip_b.z;
      w2 = w2 + 0.5f - REAL_FLOOR(w2 + 0.5f);
      real fr2 = nfft2 * w2;
      int igrid2 = REAL_FLOOR(fr2);
      w2 = fr2 - igrid2;


      real w3 = xi * recip_c.x + yi * recip_c.y + zi * recip_c.z;
      w3 = w3 + 0.5f - REAL_FLOOR(w3 + 0.5f);
      real fr3 = nfft3 * w3;
      int igrid3 = REAL_FLOOR(fr3);
      w3 = fr3 - igrid3;


      igrid1 = igrid1 - bsorder + 1;
      igrid2 = igrid2 - bsorder + 1;
      igrid3 = igrid3 - bsorder + 1;
      igrid1 += (igrid1 < 0 ? nfft1 : 0);
      igrid2 += (igrid2 < 0 ? nfft2 : 0);
      igrid3 += (igrid3 < 0 ? nfft3 : 0);

      if CONSTEXPR (WHAT == MPOLE_GRID || WHAT == UIND_GRID ||
                    WHAT == UIND_GRID_FPHI2) {
         bsplgen<4, bsorder>(w1, thetai1, array);
         bsplgen<4, bsorder>(w2, thetai2, array);
         bsplgen<4, bsorder>(w3, thetai3, array);
      }


      if CONSTEXPR (WHAT == MPOLE_GRID) {
         real tuv000 = 0;
         real tuv001 = 0;
         real tuv010 = 0;
         real tuv100 = 0;
         real tuv200 = 0;
         real tuv020 = 0;
         real tuv002 = 0;
         real tuv110 = 0;
         real tuv101 = 0;
         real tuv011 = 0;
         real tuv300 = 0;
         real tuv030 = 0;
         real tuv003 = 0;
         real tuv210 = 0;
         real tuv201 = 0;
         real tuv120 = 0;
         real tuv021 = 0;
         real tuv102 = 0;
         real tuv012 = 0;
         real tuv111 = 0;
         for (int iz = 0; iz < bsorder; ++iz) {
            int zbase = igrid3 + iz;
            zbase -= (zbase >= nfft3 ? nfft3 : 0);
            zbase *= (nfft1 * nfft2);
            real v0 = thetai3[4 * iz];
            real v1 = thetai3[4 * iz + 1];
            real v2 = thetai3[4 * iz + 2];
            real v3 = thetai3[4 * iz + 3];
            real tu00 = 0;
            real tu10 = 0;
            real tu01 = 0;
            real tu20 = 0;
            real tu11 = 0;
            real tu02 = 0;
            real tu30 = 0;
            real tu21 = 0;
            real tu12 = 0;
            real tu03 = 0;
            for (int iy = 0; iy < bsorder; ++iy) {
               int ybase = igrid2 + iy;
               ybase -= (ybase >= nfft2 ? nfft2 : 0);
               ybase *= nfft1;
               real u0 = thetai2[4 * iy];
               real u1 = thetai2[4 * iy + 1];
               real u2 = thetai2[4 * iy + 2];
               real u3 = thetai2[4 * iy + 3];
               real t0 = 0;
               real t1 = 0;
               real t2 = 0;
               real t3 = 0;
               for (int ix = 0; ix < bsorder; ++ix) {
                  int xbase = igrid1 + ix;
                  xbase -= (xbase >= nfft1 ? nfft1 : 0);
                  real tq = qgrid[2 * (xbase + ybase + zbase)];
                  t0 += tq * thetai1[4 * ix];
                  t1 += tq * thetai1[4 * ix + 1];
                  t2 += tq * thetai1[4 * ix + 2];
                  t3 += tq * thetai1[4 * ix + 3];
               }
               tu00 += t0 * u0;
               tu10 += t1 * u0;
               tu01 += t0 * u1;
               tu20 += t2 * u0;
               tu11 += t1 * u1;
               tu02 += t0 * u2;
               tu30 += t3 * u0;
               tu21 += t2 * u1;
               tu12 += t1 * u2;
               tu03 += t0 * u3;
            }
            tuv000 += tu00 * v0;
            tuv100 += tu10 * v0;
            tuv010 += tu01 * v0;
            tuv001 += tu00 * v1;
            tuv200 += tu20 * v0;
            tuv020 += tu02 * v0;
            tuv002 += tu00 * v2;
            tuv110 += tu11 * v0;
            tuv101 += tu10 * v1;
            tuv011 += tu01 * v1;
            tuv300 += tu30 * v0;
            tuv030 += tu03 * v0;
            tuv003 += tu00 * v3;
            tuv210 += tu21 * v0;
            tuv201 += tu20 * v1;
            tuv120 += tu12 * v0;
            tuv021 += tu02 * v1;
            tuv102 += tu10 * v2;
            tuv012 += tu01 * v2;
            tuv111 += tu11 * v1;
         }
         fphi[i][0] = tuv000;
         fphi[i][1] = tuv100;
         fphi[i][2] = tuv010;
         fphi[i][3] = tuv001;
         fphi[i][4] = tuv200;
         fphi[i][5] = tuv020;
         fphi[i][6] = tuv002;
         fphi[i][7] = tuv110;
         fphi[i][8] = tuv101;
         fphi[i][9] = tuv011;
         fphi[i][10] = tuv300;
         fphi[i][11] = tuv030;
         fphi[i][12] = tuv003;
         fphi[i][13] = tuv210;
         fphi[i][14] = tuv201;
         fphi[i][15] = tuv120;
         fphi[i][16] = tuv021;
         fphi[i][17] = tuv102;
         fphi[i][18] = tuv012;
         fphi[i][19] = tuv111;
      }


      if CONSTEXPR (WHAT == UIND_GRID) {
         real tuv100_1 = 0;
         real tuv010_1 = 0;
         real tuv001_1 = 0;
         real tuv200_1 = 0;
         real tuv020_1 = 0;
         real tuv002_1 = 0;
         real tuv110_1 = 0;
         real tuv101_1 = 0;
         real tuv011_1 = 0;
         real tuv100_2 = 0;
         real tuv010_2 = 0;
         real tuv001_2 = 0;
         real tuv200_2 = 0;
         real tuv020_2 = 0;
         real tuv002_2 = 0;
         real tuv110_2 = 0;
         real tuv101_2 = 0;
         real tuv011_2 = 0;
         real tuv000 = 0;
         real tuv001 = 0;
         real tuv010 = 0;
         real tuv100 = 0;
         real tuv200 = 0;
         real tuv020 = 0;
         real tuv002 = 0;
         real tuv110 = 0;
         real tuv101 = 0;
         real tuv011 = 0;
         real tuv300 = 0;
         real tuv030 = 0;
         real tuv003 = 0;
         real tuv210 = 0;
         real tuv201 = 0;
         real tuv120 = 0;
         real tuv021 = 0;
         real tuv102 = 0;
         real tuv012 = 0;
         real tuv111 = 0;
         for (int iz = 0; iz < bsorder; ++iz) {
            int zbase = igrid3 + iz;
            zbase -= (zbase >= nfft3 ? nfft3 : 0);
            zbase *= (nfft1 * nfft2);
            real v0 = thetai3[4 * iz];
            real v1 = thetai3[4 * iz + 1];
            real v2 = thetai3[4 * iz + 2];
            real v3 = thetai3[4 * iz + 3];
            real tu00_1 = 0;
            real tu01_1 = 0;
            real tu10_1 = 0;
            real tu20_1 = 0;
            real tu11_1 = 0;
            real tu02_1 = 0;
            real tu00_2 = 0;
            real tu01_2 = 0;
            real tu10_2 = 0;
            real tu20_2 = 0;
            real tu11_2 = 0;
            real tu02_2 = 0;
            real tu00 = 0;
            real tu10 = 0;
            real tu01 = 0;
            real tu20 = 0;
            real tu11 = 0;
            real tu02 = 0;
            real tu30 = 0;
            real tu21 = 0;
            real tu12 = 0;
            real tu03 = 0;
            for (int iy = 0; iy < bsorder; ++iy) {
               int ybase = igrid2 + iy;
               ybase -= (ybase >= nfft2 ? nfft2 : 0);
               ybase *= nfft1;
               real u0 = thetai2[4 * iy];
               real u1 = thetai2[4 * iy + 1];
               real u2 = thetai2[4 * iy + 2];
               real u3 = thetai2[4 * iy + 3];
               real t0_1 = 0;
               real t1_1 = 0;
               real t2_1 = 0;
               real t0_2 = 0;
               real t1_2 = 0;
               real t2_2 = 0;
               real t3 = 0;
               for (int ix = 0; ix < bsorder; ++ix) {
                  int xbase = igrid1 + ix;
                  xbase -= (xbase >= nfft1 ? nfft1 : 0);
                  real tq_1 = qgrid[2 * (xbase + ybase + zbase)];
                  real tq_2 = qgrid[2 * (xbase + ybase + zbase) + 1];
                  t0_1 += tq_1 * thetai1[4 * ix];
                  t1_1 += tq_1 * thetai1[4 * ix + 1];
                  t2_1 += tq_1 * thetai1[4 * ix + 2];
                  t0_2 += tq_2 * thetai1[4 * ix];
                  t1_2 += tq_2 * thetai1[4 * ix + 1];
                  t2_2 += tq_2 * thetai1[4 * ix + 2];
                  t3 += (tq_1 + tq_2) * thetai1[4 * ix + 3];
               }
               tu00_1 += t0_1 * u0;
               tu10_1 += t1_1 * u0;
               tu01_1 += t0_1 * u1;
               tu20_1 += t2_1 * u0;
               tu11_1 += t1_1 * u1;
               tu02_1 += t0_1 * u2;
               tu00_2 += t0_2 * u0;
               tu10_2 += t1_2 * u0;
               tu01_2 += t0_2 * u1;
               tu20_2 += t2_2 * u0;
               tu11_2 += t1_2 * u1;
               tu02_2 += t0_2 * u2;
               real t0 = t0_1 + t0_2;
               real t1 = t1_1 + t1_2;
               real t2 = t2_1 + t2_2;
               tu00 += t0 * u0;
               tu10 += t1 * u0;
               tu01 += t0 * u1;
               tu20 += t2 * u0;
               tu11 += t1 * u1;
               tu02 += t0 * u2;
               tu30 += t3 * u0;
               tu21 += t2 * u1;
               tu12 += t1 * u2;
               tu03 += t0 * u3;
            }
            tuv100_1 += tu10_1 * v0;
            tuv010_1 += tu01_1 * v0;
            tuv001_1 += tu00_1 * v1;
            tuv200_1 += tu20_1 * v0;
            tuv020_1 += tu02_1 * v0;
            tuv002_1 += tu00_1 * v2;
            tuv110_1 += tu11_1 * v0;
            tuv101_1 += tu10_1 * v1;
            tuv011_1 += tu01_1 * v1;
            tuv100_2 += tu10_2 * v0;
            tuv010_2 += tu01_2 * v0;
            tuv001_2 += tu00_2 * v1;
            tuv200_2 += tu20_2 * v0;
            tuv020_2 += tu02_2 * v0;
            tuv002_2 += tu00_2 * v2;
            tuv110_2 += tu11_2 * v0;
            tuv101_2 += tu10_2 * v1;
            tuv011_2 += tu01_2 * v1;
            tuv000 += tu00 * v0;
            tuv100 += tu10 * v0;
            tuv010 += tu01 * v0;
            tuv001 += tu00 * v1;
            tuv200 += tu20 * v0;
            tuv020 += tu02 * v0;
            tuv002 += tu00 * v2;
            tuv110 += tu11 * v0;
            tuv101 += tu10 * v1;
            tuv011 += tu01 * v1;
            tuv300 += tu30 * v0;
            tuv030 += tu03 * v0;
            tuv003 += tu00 * v3;
            tuv210 += tu21 * v0;
            tuv201 += tu20 * v1;
            tuv120 += tu12 * v0;
            tuv021 += tu02 * v1;
            tuv102 += tu10 * v2;
            tuv012 += tu01 * v2;
            tuv111 += tu11 * v1;
         } // end for (iz)
         fdip_phi1[i][0] = 0;
         fdip_phi1[i][1] = tuv100_1;
         fdip_phi1[i][2] = tuv010_1;
         fdip_phi1[i][3] = tuv001_1;
         fdip_phi1[i][4] = tuv200_1;
         fdip_phi1[i][5] = tuv020_1;
         fdip_phi1[i][6] = tuv002_1;
         fdip_phi1[i][7] = tuv110_1;
         fdip_phi1[i][8] = tuv101_1;
         fdip_phi1[i][9] = tuv011_1;
         fdip_phi2[i][0] = 0;
         fdip_phi2[i][1] = tuv100_2;
         fdip_phi2[i][2] = tuv010_2;
         fdip_phi2[i][3] = tuv001_2;
         fdip_phi2[i][4] = tuv200_2;
         fdip_phi2[i][5] = tuv020_2;
         fdip_phi2[i][6] = tuv002_2;
         fdip_phi2[i][7] = tuv110_2;
         fdip_phi2[i][8] = tuv101_2;
         fdip_phi2[i][9] = tuv011_2;
         fdip_sum_phi[i][0] = tuv000;
         fdip_sum_phi[i][1] = tuv100;
         fdip_sum_phi[i][2] = tuv010;
         fdip_sum_phi[i][3] = tuv001;
         fdip_sum_phi[i][4] = tuv200;
         fdip_sum_phi[i][5] = tuv020;
         fdip_sum_phi[i][6] = tuv002;
         fdip_sum_phi[i][7] = tuv110;
         fdip_sum_phi[i][8] = tuv101;
         fdip_sum_phi[i][9] = tuv011;
         fdip_sum_phi[i][10] = tuv300;
         fdip_sum_phi[i][11] = tuv030;
         fdip_sum_phi[i][12] = tuv003;
         fdip_sum_phi[i][13] = tuv210;
         fdip_sum_phi[i][14] = tuv201;
         fdip_sum_phi[i][15] = tuv120;
         fdip_sum_phi[i][16] = tuv021;
         fdip_sum_phi[i][17] = tuv102;
         fdip_sum_phi[i][18] = tuv012;
         fdip_sum_phi[i][19] = tuv111;
      }


      if CONSTEXPR (WHAT == UIND_GRID_FPHI2) {
         real tuv100_1 = 0;
         real tuv010_1 = 0;
         real tuv001_1 = 0;
         real tuv200_1 = 0;
         real tuv020_1 = 0;
         real tuv002_1 = 0;
         real tuv110_1 = 0;
         real tuv101_1 = 0;
         real tuv011_1 = 0;
         real tuv100_2 = 0;
         real tuv010_2 = 0;
         real tuv001_2 = 0;
         real tuv200_2 = 0;
         real tuv020_2 = 0;
         real tuv002_2 = 0;
         real tuv110_2 = 0;
         real tuv101_2 = 0;
         real tuv011_2 = 0;
         for (int iz = 0; iz < bsorder; ++iz) {
            int zbase = igrid3 + iz;
            zbase -= (zbase >= nfft3 ? nfft3 : 0);
            zbase *= (nfft1 * nfft2);
            real v0 = thetai3[4 * iz];
            real v1 = thetai3[4 * iz + 1];
            real v2 = thetai3[4 * iz + 2];
            real tu00_1 = 0;
            real tu01_1 = 0;
            real tu10_1 = 0;
            real tu20_1 = 0;
            real tu11_1 = 0;
            real tu02_1 = 0;
            real tu00_2 = 0;
            real tu01_2 = 0;
            real tu10_2 = 0;
            real tu20_2 = 0;
            real tu11_2 = 0;
            real tu02_2 = 0;
            for (int iy = 0; iy < bsorder; ++iy) {
               int ybase = igrid2 + iy;
               ybase -= (ybase >= nfft2 ? nfft2 : 0);
               ybase *= nfft1;
               real u0 = thetai2[4 * iy];
               real u1 = thetai2[4 * iy + 1];
               real u2 = thetai2[4 * iy + 2];
               real t0_1 = 0;
               real t1_1 = 0;
               real t2_1 = 0;
               real t0_2 = 0;
               real t1_2 = 0;
               real t2_2 = 0;
               for (int ix = 0; ix < bsorder; ++ix) {
                  int xbase = igrid1 + ix;
                  xbase -= (xbase >= nfft1 ? nfft1 : 0);
                  real tq_1 = qgrid[2 * (xbase + ybase + zbase)];
                  real tq_2 = qgrid[2 * (xbase + ybase + zbase) + 1];
                  t0_1 += tq_1 * thetai1[4 * ix];
                  t1_1 += tq_1 * thetai1[4 * ix + 1];
                  t2_1 += tq_1 * thetai1[4 * ix + 2];
                  t0_2 += tq_2 * thetai1[4 * ix];
                  t1_2 += tq_2 * thetai1[4 * ix + 1];
                  t2_2 += tq_2 * thetai1[4 * ix + 2];
               }
               tu00_1 += t0_1 * u0;
               tu10_1 += t1_1 * u0;
               tu01_1 += t0_1 * u1;
               tu20_1 += t2_1 * u0;
               tu11_1 += t1_1 * u1;
               tu02_1 += t0_1 * u2;
               tu00_2 += t0_2 * u0;
               tu10_2 += t1_2 * u0;
               tu01_2 += t0_2 * u1;
               tu20_2 += t2_2 * u0;
               tu11_2 += t1_2 * u1;
               tu02_2 += t0_2 * u2;
            }
            tuv100_1 += tu10_1 * v0;
            tuv010_1 += tu01_1 * v0;
            tuv001_1 += tu00_1 * v1;
            tuv200_1 += tu20_1 * v0;
            tuv020_1 += tu02_1 * v0;
            tuv002_1 += tu00_1 * v2;
            tuv110_1 += tu11_1 * v0;
            tuv101_1 += tu10_1 * v1;
            tuv011_1 += tu01_1 * v1;
            tuv100_2 += tu10_2 * v0;
            tuv010_2 += tu01_2 * v0;
            tuv001_2 += tu00_2 * v1;
            tuv200_2 += tu20_2 * v0;
            tuv020_2 += tu02_2 * v0;
            tuv002_2 += tu00_2 * v2;
            tuv110_2 += tu11_2 * v0;
            tuv101_2 += tu10_2 * v1;
            tuv011_2 += tu01_2 * v1;
         } // end for (iz)
         fdip_phi1[i][0] = 0;
         fdip_phi1[i][1] = tuv100_1;
         fdip_phi1[i][2] = tuv010_1;
         fdip_phi1[i][3] = tuv001_1;
         fdip_phi1[i][4] = tuv200_1;
         fdip_phi1[i][5] = tuv020_1;
         fdip_phi1[i][6] = tuv002_1;
         fdip_phi1[i][7] = tuv110_1;
         fdip_phi1[i][8] = tuv101_1;
         fdip_phi1[i][9] = tuv011_1;
         fdip_phi2[i][0] = 0;
         fdip_phi2[i][1] = tuv100_2;
         fdip_phi2[i][2] = tuv010_2;
         fdip_phi2[i][3] = tuv001_2;
         fdip_phi2[i][4] = tuv200_2;
         fdip_phi2[i][5] = tuv020_2;
         fdip_phi2[i][6] = tuv002_2;
         fdip_phi2[i][7] = tuv110_2;
         fdip_phi2[i][8] = tuv101_2;
         fdip_phi2[i][9] = tuv011_2;
      }
   }
}


void fphi_mpole_cu(PMEUnit pme_u, real* fphi)
{
   auto& st = *pme_u;
   int n1 = st.nfft1;
   int n2 = st.nfft2;
   int n3 = st.nfft3;


   auto ker = fphi_tmpl_cu<MPOLE_GRID, 5>;
   launch_k2s(nonblk, PME_BLOCKDIM, n, ker, n, n1, n2, n3, x, y, z, fphi,
              nullptr, nullptr, st.qgrid, recipa, recipb, recipc);
}


void fphi_uind_cu(PMEUnit pme_u, real* fdip_phi1, real* fdip_phi2,
                  real* fdip_sum_phi)
{
   auto& st = *pme_u;
   int n1 = st.nfft1;
   int n2 = st.nfft2;
   int n3 = st.nfft3;


   auto ker = fphi_tmpl_cu<UIND_GRID, 5>;
   launch_k2s(nonblk, PME_BLOCKDIM, n, ker, n, n1, n2, n3, x, y, z, fdip_phi1,
              fdip_phi2, fdip_sum_phi, st.qgrid, recipa, recipb, recipc);
}


void fphi_uind2_cu(PMEUnit pme_u, real* fdip_phi1, real* fdip_phi2)
{
   auto& st = *pme_u;
   int n1 = st.nfft1;
   int n2 = st.nfft2;
   int n3 = st.nfft3;


   auto ker = fphi_tmpl_cu<UIND_GRID_FPHI2, 5>;
   launch_k2s(nonblk, PME_BLOCKDIM, n, ker, n, n1, n2, n3, x, y, z, fdip_phi1,
              fdip_phi2, nullptr, st.qgrid, recipa, recipb, recipc);
}


void pme_cuda_func_config()
{
   // grid

   auto grid_mpole = grid_tmpl_cu<MPOLE_GRID, 5>;
   check_rt(hipFuncSetCacheConfig(reinterpret_cast<const void*>(grid_mpole), hipFuncCachePreferNone));

   auto grid_uind = grid_tmpl_cu<UIND_GRID, 5>;
   check_rt(hipFuncSetCacheConfig(reinterpret_cast<const void*>(grid_uind), hipFuncCachePreferNone));

   // fphi

   auto fphi_mpole = fphi_tmpl_cu<MPOLE_GRID, 5>;
   check_rt(hipFuncSetCacheConfig(reinterpret_cast<const void*>(fphi_mpole), hipFuncCachePreferL1));

   auto fphi_uind = fphi_tmpl_cu<UIND_GRID, 5>;
   check_rt(hipFuncSetCacheConfig(reinterpret_cast<const void*>(fphi_uind), hipFuncCachePreferL1));

   auto fphi_uind2 = fphi_tmpl_cu<UIND_GRID_FPHI2, 5>;
   check_rt(hipFuncSetCacheConfig(reinterpret_cast<const void*>(fphi_uind2), hipFuncCachePreferL1));
}
TINKER_NAMESPACE_END
