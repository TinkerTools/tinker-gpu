#include "hip/hip_runtime.h"
#include "add.h"
#include "box.h"
#include "e_mpole.h"
#include "launch.h"
#include "md.h"
#include "pme.h"
#include "pme_grid_def.h"
#include "seq_pme.h"
#include "spatial.h"


TINKER_NAMESPACE_BEGIN
static constexpr int PME_BLOCKDIM = 64;


template <class T, int bsorder>
__global__
void grid_tmpl_cu(const real* restrict x, const real* restrict y,
                  const real* restrict z, int n, int nfft1, int nfft2,
                  int nfft3, const real* restrict ptr1,
                  const real* restrict ptr2, real* restrict qgrid,
                  real3 recip_a, real3 recip_b, real3 recip_c)
{
   real thetai1[4 * 5];
   real thetai2[4 * 5];
   real thetai3[4 * 5];
   __shared__ real sharedarray[5 * 5 * PME_BLOCKDIM];
   real* restrict array = &sharedarray[5 * 5 * threadIdx.x];


   MAYBE_UNUSED const real(*fmp)[10] = (real(*)[10])ptr1;
   MAYBE_UNUSED const real(*fuind)[3] = (real(*)[3])ptr1;
   MAYBE_UNUSED const real(*fuinp)[3] = (real(*)[3])ptr2;


   for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n;
        i += blockDim.x * gridDim.x) {
      real xi = x[i];
      real yi = y[i];
      real zi = z[i];


      real w1 = xi * recip_a.x + yi * recip_a.y + zi * recip_a.z;
      w1 = w1 + 0.5f - REAL_FLOOR(w1 + 0.5f);
      real fr1 = nfft1 * w1;
      int igrid1 = REAL_FLOOR(fr1);
      w1 = fr1 - igrid1;


      real w2 = xi * recip_b.x + yi * recip_b.y + zi * recip_b.z;
      w2 = w2 + 0.5f - REAL_FLOOR(w2 + 0.5f);
      real fr2 = nfft2 * w2;
      int igrid2 = REAL_FLOOR(fr2);
      w2 = fr2 - igrid2;


      real w3 = xi * recip_c.x + yi * recip_c.y + zi * recip_c.z;
      w3 = w3 + 0.5f - REAL_FLOOR(w3 + 0.5f);
      real fr3 = nfft3 * w3;
      int igrid3 = REAL_FLOOR(fr3);
      w3 = fr3 - igrid3;


      igrid1 = igrid1 - bsorder + 1;
      igrid2 = igrid2 - bsorder + 1;
      igrid3 = igrid3 - bsorder + 1;
      igrid1 += (igrid1 < 0 ? nfft1 : 0);
      igrid2 += (igrid2 < 0 ? nfft2 : 0);
      igrid3 += (igrid3 < 0 ? nfft3 : 0);


      if CONSTEXPR (T::N == MPOLE_GRID) {
         bsplgen<3, bsorder>(w1, thetai1, array);
         bsplgen<3, bsorder>(w2, thetai2, array);
         bsplgen<3, bsorder>(w3, thetai3, array);
      }


      if CONSTEXPR (T::N == UIND_GRID) {
         bsplgen<2, bsorder>(w1, thetai1, array);
         bsplgen<2, bsorder>(w2, thetai2, array);
         bsplgen<2, bsorder>(w3, thetai3, array);
      }


      if CONSTEXPR (T::N == MPOLE_GRID) {
         real fmpi0 = fmp[i][mpl_pme_0];
         real fmpix = fmp[i][mpl_pme_x];
         real fmpiy = fmp[i][mpl_pme_y];
         real fmpiz = fmp[i][mpl_pme_z];
         real fmpixx = fmp[i][mpl_pme_xx];
         real fmpiyy = fmp[i][mpl_pme_yy];
         real fmpizz = fmp[i][mpl_pme_zz];
         real fmpixy = fmp[i][mpl_pme_xy];
         real fmpixz = fmp[i][mpl_pme_xz];
         real fmpiyz = fmp[i][mpl_pme_yz];
         for (int iz = 0; iz < bsorder; ++iz) {
            int zbase = igrid3 + iz;
            zbase -= (zbase >= nfft3 ? nfft3 : 0);
            zbase *= (nfft1 * nfft2);
            real v0 = thetai3[4 * iz];
            real v1 = thetai3[4 * iz + 1];
            real v2 = thetai3[4 * iz + 2];
            for (int iy = 0; iy < bsorder; ++iy) {
               int ybase = igrid2 + iy;
               ybase -= (ybase >= nfft2 ? nfft2 : 0);
               ybase *= nfft1;
               real u0 = thetai2[4 * iy];
               real u1 = thetai2[4 * iy + 1];
               real u2 = thetai2[4 * iy + 2];
               // fmp: 0, x, y, z, xx, yy, zz, xy, xz, yz
               //      1, 2, 3, 4,  5,  6,  7,  8,  9, 10
               real term0 = fmpi0 * u0 * v0 + fmpiy * u1 * v0 +
                  fmpiz * u0 * v1 + fmpiyy * u2 * v0 + fmpizz * u0 * v2 +
                  fmpiyz * u1 * v1;
               real term1 =
                  fmpix * u0 * v0 + fmpixy * u1 * v0 + fmpixz * u0 * v1;
               real term2 = fmpixx * u0 * v0;
               for (int ix = 0; ix < bsorder; ++ix) {
                  int xbase = igrid1 + ix;
                  xbase -= (xbase >= nfft1 ? nfft1 : 0);
                  int index = xbase + ybase + zbase;
                  real t0 = thetai1[4 * ix];
                  real t1 = thetai1[4 * ix + 1];
                  real t2 = thetai1[4 * ix + 2];
                  atomic_add(term0 * t0 + term1 * t1 + term2 * t2, qgrid,
                             2 * index);
               }
            } // end for (int iy)
         }
      } // end if (T::N == MPOLE_GRID)


      if CONSTEXPR (T::N == UIND_GRID) {
         real fuindi0 = fuind[i][0];
         real fuindi1 = fuind[i][1];
         real fuindi2 = fuind[i][2];
         real fuinpi0 = fuinp[i][0];
         real fuinpi1 = fuinp[i][1];
         real fuinpi2 = fuinp[i][2];
         #pragma unroll
         for (int iz = 0; iz < bsorder; ++iz) {
            int zbase = igrid3 + iz;
            zbase -= (zbase >= nfft3 ? nfft3 : 0);
            zbase *= (nfft1 * nfft2);
            real v0 = thetai3[4 * iz];
            real v1 = thetai3[4 * iz + 1];
            #pragma unroll
            for (int iy = 0; iy < bsorder; ++iy) {
               int ybase = igrid2 + iy;
               ybase -= (ybase >= nfft2 ? nfft2 : 0);
               ybase *= nfft1;
               real u0 = thetai2[4 * iy];
               real u1 = thetai2[4 * iy + 1];
               real term01 = fuindi1 * u1 * v0 + fuindi2 * u0 * v1;
               real term11 = fuindi0 * u0 * v0;
               real term02 = fuinpi1 * u1 * v0 + fuinpi2 * u0 * v1;
               real term12 = fuinpi0 * u0 * v0;
               #pragma unroll
               for (int ix = 0; ix < bsorder; ++ix) {
                  int xbase = igrid1 + ix;
                  xbase -= (xbase >= nfft1 ? nfft1 : 0);
                  int index = xbase + ybase + zbase;
                  real t0 = thetai1[4 * ix];
                  real t1 = thetai1[4 * ix + 1];
                  atomic_add(term01 * t0 + term11 * t1, qgrid, 2 * index);
                  atomic_add(term02 * t0 + term12 * t1, qgrid, 2 * index + 1);
               }
            } // end for (int iy)
         }
      } // end if (T::N == UIND_GRID)
   }
}

namespace platform {
namespace cu {
template <class T, int bsorder>
__global__
void grid_put(const int* restrict igrid, const real* restrict thetai1,
              const real* restrict thetai2, const real* restrict thetai3,
              const Spatial::SortedAtom* restrict sorted, int n, int padded_n,
              int nfft1, int nfft2, int nfft3, const real* restrict ptr1,
              const real* ptr2, real* restrict qgrid)
{
   constexpr int bso2 = bsorder * bsorder;
   constexpr int bso3 = bsorder * bso2;
   for (int m = threadIdx.x + blockIdx.x * blockDim.x; m < n * bso3;
        m += blockDim.x * gridDim.x) {
      // m = i0 * bso3 + j;
      int i0 = m / bso3;
      int j = m - i0 * bso3;


      int i = sorted[i0].unsorted;
      int igrid1 = igrid[3 * i + 0];
      int igrid2 = igrid[3 * i + 1];
      int igrid3 = igrid[3 * i + 2];


      int iz = j / bso2;
      j -= iz * bso2;
      int iy = j / bsorder;
      int ix = j - (j / bsorder) * bsorder;


      real v0 = thetai3[(4 * iz + 0) * padded_n + i];
      real v1 = thetai3[(4 * iz + 1) * padded_n + i];
      int zbase = igrid3 + iz;
      zbase -= (zbase >= nfft3 ? nfft3 : 0);
      zbase *= (nfft1 * nfft2);


      real u0 = thetai2[(4 * iy + 0) * padded_n + i];
      real u1 = thetai2[(4 * iy + 1) * padded_n + i];
      int ybase = igrid2 + iy;
      ybase -= (ybase >= nfft2 ? nfft2 : 0);
      ybase *= nfft1;


      real t0 = thetai1[(4 * ix + 0) * padded_n + i];
      real t1 = thetai1[(4 * ix + 1) * padded_n + i];
      int xbase = igrid1 + ix;
      xbase -= (xbase >= nfft1 ? nfft1 : 0);
      int index = xbase + ybase + zbase;


      if CONSTEXPR (T::N == MPOLE_GRID) {
         real v2 = thetai3[(4 * iz + 2) * padded_n + i];
         real u2 = thetai2[(4 * iy + 2) * padded_n + i];
         real t2 = thetai1[(4 * ix + 2) * padded_n + i];
         real fmpi0 = ptr1[i * 10 + mpl_pme_0];
         real fmpix = ptr1[i * 10 + mpl_pme_x];
         real fmpiy = ptr1[i * 10 + mpl_pme_y];
         real fmpiz = ptr1[i * 10 + mpl_pme_z];
         real fmpixx = ptr1[i * 10 + mpl_pme_xx];
         real fmpiyy = ptr1[i * 10 + mpl_pme_yy];
         real fmpizz = ptr1[i * 10 + mpl_pme_zz];
         real fmpixy = ptr1[i * 10 + mpl_pme_xy];
         real fmpixz = ptr1[i * 10 + mpl_pme_xz];
         real fmpiyz = ptr1[i * 10 + mpl_pme_yz];
         real term0 = fmpi0 * u0 * v0 + fmpiy * u1 * v0 + fmpiz * u0 * v1 +
            fmpiyy * u2 * v0 + fmpizz * u0 * v2 + fmpiyz * u1 * v1;
         real term1 = fmpix * u0 * v0 + fmpixy * u1 * v0 + fmpixz * u0 * v1;
         real term2 = fmpixx * u0 * v0;
         atomic_add(term0 * t0 + term1 * t1 + term2 * t2, qgrid, 2 * index);
      }


      if CONSTEXPR (T::N == UIND_GRID) {
         real3 fd =
            make_real3(ptr1[3 * i + 0], ptr1[3 * i + 1], ptr1[3 * i + 2]);
         real3 fp =
            make_real3(ptr2[3 * i + 0], ptr2[3 * i + 1], ptr2[3 * i + 2]);
         real3 tuv = make_real3(t1 * u0 * v0, t0 * u1 * v0, t0 * u0 * v1);
         atomic_add(dot3(fd, tuv), qgrid, 2 * index);
         atomic_add(dot3(fp, tuv), qgrid, 2 * index + 1);
      }
   }
}


void grid_mpole(PMEUnit pme_u, real (*fmp)[10])
{
   auto& st = *pme_u;
   int n1 = st.nfft1;
   int n2 = st.nfft2;
   int n3 = st.nfft3;
   int nt = n1 * n2 * n3;


   device_array::zero(false, 2 * nt, st.qgrid);
#if 1
   auto ker = grid_tmpl_cu<MPOLE, 5>;
   launch_k2s(nonblk, PME_BLOCKDIM, n, ker, x, y, z, n, n1, n2, n3,
              (const real*)fmp, nullptr, st.qgrid, recipa, recipb, recipc);
#elif 0
   auto ker = grid_put<MPOLE, 5>;
   int npa = 5 * 5 * 5 * n;
   launch_k1s(nonblk, npa, ker, st.igrid, st.thetai1, st.thetai2, st.thetai3,
              mspatial_unit->sorted, n, padded_n, n1, n2, n3, (const real*)fmp,
              nullptr, st.qgrid);
#endif
}


void grid_uind(PMEUnit pme_u, real (*fuind)[3], real (*fuinp)[3])
{
   auto& st = *pme_u;
   int n1 = st.nfft1;
   int n2 = st.nfft2;
   int n3 = st.nfft3;
   int nt = n1 * n2 * n3;


   device_array::zero(false, 2 * nt, st.qgrid);
#if 0
   auto ker = grid_tmpl_cu<UIND, 5>;
   launch_k2s(nonblk, PME_BLOCKDIM, n, ker, x, y, z, n, n1, n2, n3,
              (const real*)fuind, (const real*)fuinp, st.qgrid, recipa, recipb,
              recipc);
#elif 1
   auto ker = grid_put<UIND, 5>;
   int npa = 5 * 5 * 5 * n;
   launch_k1s(nonblk, npa, ker, st.igrid, st.thetai1, st.thetai2, st.thetai3,
              mspatial_unit->sorted, n, padded_n, n1, n2, n3,
              (const real*)fuind, (const real*)fuinp, st.qgrid);
#endif
}


template <int LEVEL, int bsorder>
__global__
void bspline_fill(int* restrict igrid, real* restrict thetai1,
                  real* restrict thetai2, real* restrict thetai3,
                  const real* restrict x, const real* restrict y,
                  const real* restrict z, int n, int padded_n, int nfft1,
                  int nfft2, int nfft3, real3 recip_a, real3 recip_b,
                  real3 recip_c)
{
   const int nfft4[3] = {nfft1, nfft2, nfft3};
   const real3 recip4[3] = {recip_a, recip_b, recip_c};
   real* const thetai[3] = {thetai1, thetai2, thetai3};
   real array[5 * 5];


   for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n;
        i += blockDim.x * gridDim.x) {
      real xi = x[i];
      real yi = y[i];
      real zi = z[i];
      int igridi[3];
      for (int j = 0; j < 3; ++j) {
         real w4 = xi * recip4[j].x + yi * recip4[j].y + zi * recip4[j].z;
         w4 = w4 + 0.5f - REAL_FLOOR(w4 + 0.5f);
         real fr4 = nfft4[j] * w4;
         int igrid4 = REAL_FLOOR(fr4);
         w4 = fr4 - igrid4;
         igrid4 = igrid4 - bsorder + 1;
         igrid4 += (igrid4 < 0 ? nfft4[j] : 0);
         // write output
         igridi[j] = igrid4;
         bsplgen2<LEVEL, bsorder>(w4, thetai[j], i, padded_n, array);
      }
      igrid[3 * i + 0] = igridi[0];
      igrid[3 * i + 1] = igridi[1];
      igrid[3 * i + 2] = igridi[2];
   }
}
}
}


void bspline_fill(PMEUnit u, int level)
{
   auto& st = *u;
   if (level == 2) {
      auto ker = platform::cu::bspline_fill<2, 5>;
      launch_k1s(nonblk, n, ker, st.igrid, st.thetai1, st.thetai2, st.thetai3,
                 x, y, z, n, padded_n, st.nfft1, st.nfft2, st.nfft3, recipa,
                 recipb, recipc);
   } else if (level == 3) {
      auto ker = platform::cu::bspline_fill<3, 5>;
      launch_k1s(nonblk, n, ker, st.igrid, st.thetai1, st.thetai2, st.thetai3,
                 x, y, z, n, padded_n, st.nfft1, st.nfft2, st.nfft3, recipa,
                 recipb, recipc);
   }
}


template <class T, int bsorder>
__global__
void fphi_tmpl_cu(int n, int nfft1, int nfft2, int nfft3,
                  const real* restrict x, const real* restrict y,
                  const real* restrict z, real* restrict opt1,
                  real* restrict opt2, real* restrict opt3,
                  const real* restrict qgrid, real3 recip_a, real3 recip_b,
                  real3 recip_c)
{
   real thetai1[4 * 5];
   real thetai2[4 * 5];
   real thetai3[4 * 5];
   real array[5 * 5];


   MAYBE_UNUSED real(*restrict fphi)[20] = (real(*)[20])opt1;
   MAYBE_UNUSED real(*restrict fdip_phi1)[10] = (real(*)[10])opt1;
   MAYBE_UNUSED real(*restrict fdip_phi2)[10] = (real(*)[10])opt2;
   MAYBE_UNUSED real(*restrict fdip_sum_phi)[20] = (real(*)[20])opt3;


   for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n;
        i += blockDim.x * gridDim.x) {
      real xi = x[i];
      real yi = y[i];
      real zi = z[i];


      real w1 = xi * recip_a.x + yi * recip_a.y + zi * recip_a.z;
      w1 = w1 + 0.5f - REAL_FLOOR(w1 + 0.5f);
      real fr1 = nfft1 * w1;
      int igrid1 = REAL_FLOOR(fr1);
      w1 = fr1 - igrid1;


      real w2 = xi * xi * recip_b.x + yi * recip_b.y + zi * recip_b.z;
      w2 = w2 + 0.5f - REAL_FLOOR(w2 + 0.5f);
      real fr2 = nfft2 * w2;
      int igrid2 = REAL_FLOOR(fr2);
      w2 = fr2 - igrid2;


      real w3 = xi * recip_c.x + yi * recip_c.y + zi * recip_c.z;
      w3 = w3 + 0.5f - REAL_FLOOR(w3 + 0.5f);
      real fr3 = nfft3 * w3;
      int igrid3 = REAL_FLOOR(fr3);
      w3 = fr3 - igrid3;


      igrid1 = igrid1 - bsorder + 1;
      igrid2 = igrid2 - bsorder + 1;
      igrid3 = igrid3 - bsorder + 1;
      igrid1 += (igrid1 < 0 ? nfft1 : 0);
      igrid2 += (igrid2 < 0 ? nfft2 : 0);
      igrid3 += (igrid3 < 0 ? nfft3 : 0);

      if CONSTEXPR (T::N == MPOLE_GRID || T::N == UIND_GRID ||
                    T::N == UIND_GRID_FPHI2) {
         bsplgen<4, bsorder>(w1, thetai1, array);
         bsplgen<4, bsorder>(w2, thetai2, array);
         bsplgen<4, bsorder>(w3, thetai3, array);
      }


      if CONSTEXPR (T::N == MPOLE_GRID) {
         real tuv000 = 0;
         real tuv001 = 0;
         real tuv010 = 0;
         real tuv100 = 0;
         real tuv200 = 0;
         real tuv020 = 0;
         real tuv002 = 0;
         real tuv110 = 0;
         real tuv101 = 0;
         real tuv011 = 0;
         real tuv300 = 0;
         real tuv030 = 0;
         real tuv003 = 0;
         real tuv210 = 0;
         real tuv201 = 0;
         real tuv120 = 0;
         real tuv021 = 0;
         real tuv102 = 0;
         real tuv012 = 0;
         real tuv111 = 0;
         for (int iz = 0; iz < bsorder; ++iz) {
            int zbase = igrid3 + iz;
            zbase -= (zbase >= nfft3 ? nfft3 : 0);
            zbase *= (nfft1 * nfft2);
            real v0 = thetai3[4 * iz];
            real v1 = thetai3[4 * iz + 1];
            real v2 = thetai3[4 * iz + 2];
            real v3 = thetai3[4 * iz + 3];
            real tu00 = 0;
            real tu10 = 0;
            real tu01 = 0;
            real tu20 = 0;
            real tu11 = 0;
            real tu02 = 0;
            real tu30 = 0;
            real tu21 = 0;
            real tu12 = 0;
            real tu03 = 0;
            for (int iy = 0; iy < bsorder; ++iy) {
               int ybase = igrid2 + iy;
               ybase -= (ybase >= nfft2 ? nfft2 : 0);
               ybase *= nfft1;
               real u0 = thetai2[4 * iy];
               real u1 = thetai2[4 * iy + 1];
               real u2 = thetai2[4 * iy + 2];
               real u3 = thetai2[4 * iy + 3];
               real t0 = 0;
               real t1 = 0;
               real t2 = 0;
               real t3 = 0;
               for (int ix = 0; ix < bsorder; ++ix) {
                  int xbase = igrid1 + ix;
                  xbase -= (xbase >= nfft1 ? nfft1 : 0);
                  real tq = qgrid[2 * (xbase + ybase + zbase)];
                  t0 += tq * thetai1[4 * ix];
                  t1 += tq * thetai1[4 * ix + 1];
                  t2 += tq * thetai1[4 * ix + 2];
                  t3 += tq * thetai1[4 * ix + 3];
               }
               tu00 += t0 * u0;
               tu10 += t1 * u0;
               tu01 += t0 * u1;
               tu20 += t2 * u0;
               tu11 += t1 * u1;
               tu02 += t0 * u2;
               tu30 += t3 * u0;
               tu21 += t2 * u1;
               tu12 += t1 * u2;
               tu03 += t0 * u3;
            }
            tuv000 += tu00 * v0;
            tuv100 += tu10 * v0;
            tuv010 += tu01 * v0;
            tuv001 += tu00 * v1;
            tuv200 += tu20 * v0;
            tuv020 += tu02 * v0;
            tuv002 += tu00 * v2;
            tuv110 += tu11 * v0;
            tuv101 += tu10 * v1;
            tuv011 += tu01 * v1;
            tuv300 += tu30 * v0;
            tuv030 += tu03 * v0;
            tuv003 += tu00 * v3;
            tuv210 += tu21 * v0;
            tuv201 += tu20 * v1;
            tuv120 += tu12 * v0;
            tuv021 += tu02 * v1;
            tuv102 += tu10 * v2;
            tuv012 += tu01 * v2;
            tuv111 += tu11 * v1;
         }
         fphi[i][0] = tuv000;
         fphi[i][1] = tuv100;
         fphi[i][2] = tuv010;
         fphi[i][3] = tuv001;
         fphi[i][4] = tuv200;
         fphi[i][5] = tuv020;
         fphi[i][6] = tuv002;
         fphi[i][7] = tuv110;
         fphi[i][8] = tuv101;
         fphi[i][9] = tuv011;
         fphi[i][10] = tuv300;
         fphi[i][11] = tuv030;
         fphi[i][12] = tuv003;
         fphi[i][13] = tuv210;
         fphi[i][14] = tuv201;
         fphi[i][15] = tuv120;
         fphi[i][16] = tuv021;
         fphi[i][17] = tuv102;
         fphi[i][18] = tuv012;
         fphi[i][19] = tuv111;
      }


      if CONSTEXPR (T::N == UIND_GRID) {
         real tuv100_1 = 0;
         real tuv010_1 = 0;
         real tuv001_1 = 0;
         real tuv200_1 = 0;
         real tuv020_1 = 0;
         real tuv002_1 = 0;
         real tuv110_1 = 0;
         real tuv101_1 = 0;
         real tuv011_1 = 0;
         real tuv100_2 = 0;
         real tuv010_2 = 0;
         real tuv001_2 = 0;
         real tuv200_2 = 0;
         real tuv020_2 = 0;
         real tuv002_2 = 0;
         real tuv110_2 = 0;
         real tuv101_2 = 0;
         real tuv011_2 = 0;
         real tuv000 = 0;
         real tuv001 = 0;
         real tuv010 = 0;
         real tuv100 = 0;
         real tuv200 = 0;
         real tuv020 = 0;
         real tuv002 = 0;
         real tuv110 = 0;
         real tuv101 = 0;
         real tuv011 = 0;
         real tuv300 = 0;
         real tuv030 = 0;
         real tuv003 = 0;
         real tuv210 = 0;
         real tuv201 = 0;
         real tuv120 = 0;
         real tuv021 = 0;
         real tuv102 = 0;
         real tuv012 = 0;
         real tuv111 = 0;
         for (int iz = 0; iz < bsorder; ++iz) {
            int zbase = igrid3 + iz;
            zbase -= (zbase >= nfft3 ? nfft3 : 0);
            zbase *= (nfft1 * nfft2);
            real v0 = thetai3[4 * iz];
            real v1 = thetai3[4 * iz + 1];
            real v2 = thetai3[4 * iz + 2];
            real v3 = thetai3[4 * iz + 3];
            real tu00_1 = 0;
            real tu01_1 = 0;
            real tu10_1 = 0;
            real tu20_1 = 0;
            real tu11_1 = 0;
            real tu02_1 = 0;
            real tu00_2 = 0;
            real tu01_2 = 0;
            real tu10_2 = 0;
            real tu20_2 = 0;
            real tu11_2 = 0;
            real tu02_2 = 0;
            real tu00 = 0;
            real tu10 = 0;
            real tu01 = 0;
            real tu20 = 0;
            real tu11 = 0;
            real tu02 = 0;
            real tu30 = 0;
            real tu21 = 0;
            real tu12 = 0;
            real tu03 = 0;
            for (int iy = 0; iy < bsorder; ++iy) {
               int ybase = igrid2 + iy;
               ybase -= (ybase >= nfft2 ? nfft2 : 0);
               ybase *= nfft1;
               real u0 = thetai2[4 * iy];
               real u1 = thetai2[4 * iy + 1];
               real u2 = thetai2[4 * iy + 2];
               real u3 = thetai2[4 * iy + 3];
               real t0_1 = 0;
               real t1_1 = 0;
               real t2_1 = 0;
               real t0_2 = 0;
               real t1_2 = 0;
               real t2_2 = 0;
               real t3 = 0;
               for (int ix = 0; ix < bsorder; ++ix) {
                  int xbase = igrid1 + ix;
                  xbase -= (xbase >= nfft1 ? nfft1 : 0);
                  real tq_1 = qgrid[2 * (xbase + ybase + zbase)];
                  real tq_2 = qgrid[2 * (xbase + ybase + zbase) + 1];
                  t0_1 += tq_1 * thetai1[4 * ix];
                  t1_1 += tq_1 * thetai1[4 * ix + 1];
                  t2_1 += tq_1 * thetai1[4 * ix + 2];
                  t0_2 += tq_2 * thetai1[4 * ix];
                  t1_2 += tq_2 * thetai1[4 * ix + 1];
                  t2_2 += tq_2 * thetai1[4 * ix + 2];
                  t3 += (tq_1 + tq_2) * thetai1[4 * ix + 3];
               }
               tu00_1 += t0_1 * u0;
               tu10_1 += t1_1 * u0;
               tu01_1 += t0_1 * u1;
               tu20_1 += t2_1 * u0;
               tu11_1 += t1_1 * u1;
               tu02_1 += t0_1 * u2;
               tu00_2 += t0_2 * u0;
               tu10_2 += t1_2 * u0;
               tu01_2 += t0_2 * u1;
               tu20_2 += t2_2 * u0;
               tu11_2 += t1_2 * u1;
               tu02_2 += t0_2 * u2;
               real t0 = t0_1 + t0_2;
               real t1 = t1_1 + t1_2;
               real t2 = t2_1 + t2_2;
               tu00 += t0 * u0;
               tu10 += t1 * u0;
               tu01 += t0 * u1;
               tu20 += t2 * u0;
               tu11 += t1 * u1;
               tu02 += t0 * u2;
               tu30 += t3 * u0;
               tu21 += t2 * u1;
               tu12 += t1 * u2;
               tu03 += t0 * u3;
            }
            tuv100_1 += tu10_1 * v0;
            tuv010_1 += tu01_1 * v0;
            tuv001_1 += tu00_1 * v1;
            tuv200_1 += tu20_1 * v0;
            tuv020_1 += tu02_1 * v0;
            tuv002_1 += tu00_1 * v2;
            tuv110_1 += tu11_1 * v0;
            tuv101_1 += tu10_1 * v1;
            tuv011_1 += tu01_1 * v1;
            tuv100_2 += tu10_2 * v0;
            tuv010_2 += tu01_2 * v0;
            tuv001_2 += tu00_2 * v1;
            tuv200_2 += tu20_2 * v0;
            tuv020_2 += tu02_2 * v0;
            tuv002_2 += tu00_2 * v2;
            tuv110_2 += tu11_2 * v0;
            tuv101_2 += tu10_2 * v1;
            tuv011_2 += tu01_2 * v1;
            tuv000 += tu00 * v0;
            tuv100 += tu10 * v0;
            tuv010 += tu01 * v0;
            tuv001 += tu00 * v1;
            tuv200 += tu20 * v0;
            tuv020 += tu02 * v0;
            tuv002 += tu00 * v2;
            tuv110 += tu11 * v0;
            tuv101 += tu10 * v1;
            tuv011 += tu01 * v1;
            tuv300 += tu30 * v0;
            tuv030 += tu03 * v0;
            tuv003 += tu00 * v3;
            tuv210 += tu21 * v0;
            tuv201 += tu20 * v1;
            tuv120 += tu12 * v0;
            tuv021 += tu02 * v1;
            tuv102 += tu10 * v2;
            tuv012 += tu01 * v2;
            tuv111 += tu11 * v1;
         } // end for (iz)
         fdip_phi1[i][0] = 0;
         fdip_phi1[i][1] = tuv100_1;
         fdip_phi1[i][2] = tuv010_1;
         fdip_phi1[i][3] = tuv001_1;
         fdip_phi1[i][4] = tuv200_1;
         fdip_phi1[i][5] = tuv020_1;
         fdip_phi1[i][6] = tuv002_1;
         fdip_phi1[i][7] = tuv110_1;
         fdip_phi1[i][8] = tuv101_1;
         fdip_phi1[i][9] = tuv011_1;
         fdip_phi2[i][0] = 0;
         fdip_phi2[i][1] = tuv100_2;
         fdip_phi2[i][2] = tuv010_2;
         fdip_phi2[i][3] = tuv001_2;
         fdip_phi2[i][4] = tuv200_2;
         fdip_phi2[i][5] = tuv020_2;
         fdip_phi2[i][6] = tuv002_2;
         fdip_phi2[i][7] = tuv110_2;
         fdip_phi2[i][8] = tuv101_2;
         fdip_phi2[i][9] = tuv011_2;
         fdip_sum_phi[i][0] = tuv000;
         fdip_sum_phi[i][1] = tuv100;
         fdip_sum_phi[i][2] = tuv010;
         fdip_sum_phi[i][3] = tuv001;
         fdip_sum_phi[i][4] = tuv200;
         fdip_sum_phi[i][5] = tuv020;
         fdip_sum_phi[i][6] = tuv002;
         fdip_sum_phi[i][7] = tuv110;
         fdip_sum_phi[i][8] = tuv101;
         fdip_sum_phi[i][9] = tuv011;
         fdip_sum_phi[i][10] = tuv300;
         fdip_sum_phi[i][11] = tuv030;
         fdip_sum_phi[i][12] = tuv003;
         fdip_sum_phi[i][13] = tuv210;
         fdip_sum_phi[i][14] = tuv201;
         fdip_sum_phi[i][15] = tuv120;
         fdip_sum_phi[i][16] = tuv021;
         fdip_sum_phi[i][17] = tuv102;
         fdip_sum_phi[i][18] = tuv012;
         fdip_sum_phi[i][19] = tuv111;
      }


      if CONSTEXPR (T::N == UIND_GRID_FPHI2) {
         real tuv100_1 = 0;
         real tuv010_1 = 0;
         real tuv001_1 = 0;
         real tuv200_1 = 0;
         real tuv020_1 = 0;
         real tuv002_1 = 0;
         real tuv110_1 = 0;
         real tuv101_1 = 0;
         real tuv011_1 = 0;
         real tuv100_2 = 0;
         real tuv010_2 = 0;
         real tuv001_2 = 0;
         real tuv200_2 = 0;
         real tuv020_2 = 0;
         real tuv002_2 = 0;
         real tuv110_2 = 0;
         real tuv101_2 = 0;
         real tuv011_2 = 0;
         for (int iz = 0; iz < bsorder; ++iz) {
            int zbase = igrid3 + iz;
            zbase -= (zbase >= nfft3 ? nfft3 : 0);
            zbase *= (nfft1 * nfft2);
            real v0 = thetai3[4 * iz];
            real v1 = thetai3[4 * iz + 1];
            real v2 = thetai3[4 * iz + 2];
            real tu00_1 = 0;
            real tu01_1 = 0;
            real tu10_1 = 0;
            real tu20_1 = 0;
            real tu11_1 = 0;
            real tu02_1 = 0;
            real tu00_2 = 0;
            real tu01_2 = 0;
            real tu10_2 = 0;
            real tu20_2 = 0;
            real tu11_2 = 0;
            real tu02_2 = 0;
            for (int iy = 0; iy < bsorder; ++iy) {
               int ybase = igrid2 + iy;
               ybase -= (ybase >= nfft2 ? nfft2 : 0);
               ybase *= nfft1;
               real u0 = thetai2[4 * iy];
               real u1 = thetai2[4 * iy + 1];
               real u2 = thetai2[4 * iy + 2];
               real t0_1 = 0;
               real t1_1 = 0;
               real t2_1 = 0;
               real t0_2 = 0;
               real t1_2 = 0;
               real t2_2 = 0;
               for (int ix = 0; ix < bsorder; ++ix) {
                  int xbase = igrid1 + ix;
                  xbase -= (xbase >= nfft1 ? nfft1 : 0);
                  real tq_1 = qgrid[2 * (xbase + ybase + zbase)];
                  real tq_2 = qgrid[2 * (xbase + ybase + zbase) + 1];
                  t0_1 += tq_1 * thetai1[4 * ix];
                  t1_1 += tq_1 * thetai1[4 * ix + 1];
                  t2_1 += tq_1 * thetai1[4 * ix + 2];
                  t0_2 += tq_2 * thetai1[4 * ix];
                  t1_2 += tq_2 * thetai1[4 * ix + 1];
                  t2_2 += tq_2 * thetai1[4 * ix + 2];
               }
               tu00_1 += t0_1 * u0;
               tu10_1 += t1_1 * u0;
               tu01_1 += t0_1 * u1;
               tu20_1 += t2_1 * u0;
               tu11_1 += t1_1 * u1;
               tu02_1 += t0_1 * u2;
               tu00_2 += t0_2 * u0;
               tu10_2 += t1_2 * u0;
               tu01_2 += t0_2 * u1;
               tu20_2 += t2_2 * u0;
               tu11_2 += t1_2 * u1;
               tu02_2 += t0_2 * u2;
            }
            tuv100_1 += tu10_1 * v0;
            tuv010_1 += tu01_1 * v0;
            tuv001_1 += tu00_1 * v1;
            tuv200_1 += tu20_1 * v0;
            tuv020_1 += tu02_1 * v0;
            tuv002_1 += tu00_1 * v2;
            tuv110_1 += tu11_1 * v0;
            tuv101_1 += tu10_1 * v1;
            tuv011_1 += tu01_1 * v1;
            tuv100_2 += tu10_2 * v0;
            tuv010_2 += tu01_2 * v0;
            tuv001_2 += tu00_2 * v1;
            tuv200_2 += tu20_2 * v0;
            tuv020_2 += tu02_2 * v0;
            tuv002_2 += tu00_2 * v2;
            tuv110_2 += tu11_2 * v0;
            tuv101_2 += tu10_2 * v1;
            tuv011_2 += tu01_2 * v1;
         } // end for (iz)
         fdip_phi1[i][0] = 0;
         fdip_phi1[i][1] = tuv100_1;
         fdip_phi1[i][2] = tuv010_1;
         fdip_phi1[i][3] = tuv001_1;
         fdip_phi1[i][4] = tuv200_1;
         fdip_phi1[i][5] = tuv020_1;
         fdip_phi1[i][6] = tuv002_1;
         fdip_phi1[i][7] = tuv110_1;
         fdip_phi1[i][8] = tuv101_1;
         fdip_phi1[i][9] = tuv011_1;
         fdip_phi2[i][0] = 0;
         fdip_phi2[i][1] = tuv100_2;
         fdip_phi2[i][2] = tuv010_2;
         fdip_phi2[i][3] = tuv001_2;
         fdip_phi2[i][4] = tuv200_2;
         fdip_phi2[i][5] = tuv020_2;
         fdip_phi2[i][6] = tuv002_2;
         fdip_phi2[i][7] = tuv110_2;
         fdip_phi2[i][8] = tuv101_2;
         fdip_phi2[i][9] = tuv011_2;
      }
   }
}


void fphi_mpole_cu(PMEUnit pme_u, real* fphi)
{
   auto& st = *pme_u;
   int n1 = st.nfft1;
   int n2 = st.nfft2;
   int n3 = st.nfft3;


   auto ker = fphi_tmpl_cu<MPOLE, 5>;
   launch_k2s(nonblk, PME_BLOCKDIM, n, ker, n, n1, n2, n3, x, y, z, fphi,
              nullptr, nullptr, st.qgrid, recipa, recipb, recipc);
}


void fphi_uind_cu(PMEUnit pme_u, real* fdip_phi1, real* fdip_phi2,
                  real* fdip_sum_phi)
{
   auto& st = *pme_u;
   int n1 = st.nfft1;
   int n2 = st.nfft2;
   int n3 = st.nfft3;


   auto ker = fphi_tmpl_cu<UIND, 5>;
   launch_k2s(nonblk, PME_BLOCKDIM, n, ker, n, n1, n2, n3, x, y, z, fdip_phi1,
              fdip_phi2, fdip_sum_phi, st.qgrid, recipa, recipb, recipc);
}


void fphi_uind2_cu(PMEUnit pme_u, real* fdip_phi1, real* fdip_phi2)
{
   auto& st = *pme_u;
   int n1 = st.nfft1;
   int n2 = st.nfft2;
   int n3 = st.nfft3;


   auto ker = fphi_tmpl_cu<UIND2, 5>;
   launch_k2s(nonblk, PME_BLOCKDIM, n, ker, n, n1, n2, n3, x, y, z, fdip_phi1,
              fdip_phi2, nullptr, st.qgrid, recipa, recipb, recipc);
}


void pme_cuda_func_config()
{
   // grid

   auto grid_mpolek = grid_tmpl_cu<MPOLE, 5>;
   check_rt(hipFuncSetCacheConfig(reinterpret_cast<const void*>(grid_mpolek), hipFuncCachePreferNone));

   auto grid_uindk = grid_tmpl_cu<UIND, 5>;
   check_rt(hipFuncSetCacheConfig(reinterpret_cast<const void*>(grid_uindk), hipFuncCachePreferNone));

   // fphi

   auto fphi_mpole = fphi_tmpl_cu<MPOLE, 5>;
   check_rt(hipFuncSetCacheConfig(reinterpret_cast<const void*>(fphi_mpole), hipFuncCachePreferL1));

   auto fphi_uind = fphi_tmpl_cu<UIND, 5>;
   check_rt(hipFuncSetCacheConfig(reinterpret_cast<const void*>(fphi_uind), hipFuncCachePreferL1));

   auto fphi_uind2 = fphi_tmpl_cu<UIND2, 5>;
   check_rt(hipFuncSetCacheConfig(reinterpret_cast<const void*>(fphi_uind2), hipFuncCachePreferL1));
}
TINKER_NAMESPACE_END
