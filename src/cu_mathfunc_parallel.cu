#include "cudalib.h"
#include "error.h"
#include "mathfunc_parallel_cu.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>


TINKER_NAMESPACE_BEGIN
namespace platform {
namespace cu {
template <>
void dotprod<float>(float* ans, const float* a, const float* b, int nelem,
                    int sync)
{
   hipblasHandle_t hd = (sync ? h_cublas : h_cublas_nonblk);
   float alpha = 1, beta = 0;
   check_rt(hipblasSgemm(hd, HIPBLAS_OP_N, HIPBLAS_OP_T, 1, 1, nelem, //
                        &alpha, a, 1, b, 1,                        //
                        &beta, ans, 1));
   if (sync)
      check_rt(hipStreamSynchronize(nullptr));
}


template <>
void dotprod<double>(double* ans, const double* a, const double* b, int nelem,
                     int sync)
{
   hipblasHandle_t hd = (sync ? h_cublas : h_cublas_nonblk);
   double alpha = 1, beta = 0;
   check_rt(hipblasDgemm(hd, HIPBLAS_OP_N, HIPBLAS_OP_T, 1, 1, nelem, //
                        &alpha, a, 1, b, 1,                        //
                        &beta, ans, 1));
   if (sync)
      check_rt(hipStreamSynchronize(nullptr));
}
}
}
TINKER_NAMESPACE_END
