#include "hip/hip_runtime.h"
#include "add.h"
#include "e_mpole.h"
#include "empole_self.cuh"
#include "launch.cuh"
#include "md.h"
#include "pme.h"
#include "seq_image.h"
#include "seq_pair_mpole.h"
#include "spatial.h"


TINKER_NAMESPACE_BEGIN
#define EMPOLE_ARGS                                                            \
   size_t bufsize, count_buffer restrict nem, energy_buffer restrict em,       \
      virial_buffer restrict vir_em, real *restrict gx, real *restrict gy,     \
      real *restrict gz, real *restrict trqx, real *restrict trqy,             \
      real *restrict trqz, const Box *restrict box, real off2, real f,         \
      const real(*restrict rpole)[10]


template <int USE, elec_t ETYP>
__global__
void empole_cu1(EMPOLE_ARGS, const Spatial::SortedAtom* restrict sorted,
                int niak, const int* restrict iak, const int* restrict lst,
                int n, real aewald)
{
   constexpr int do_e = USE & calc::energy;
   constexpr int do_a = USE & calc::analyz;
   constexpr int do_g = USE & calc::grad;
   constexpr int do_v = USE & calc::virial;


   const int ithread = threadIdx.x + blockIdx.x * blockDim.x;
   const int iwarp = ithread / WARP_SIZE;
   const int nwarp = blockDim.x * gridDim.x / WARP_SIZE;
   const int ilane = threadIdx.x & (WARP_SIZE - 1);
   const int offset = ithread & (bufsize - 1);


   MAYBE_UNUSED int ctl;
   MAYBE_UNUSED real etl;
   MAYBE_UNUSED real vtlxx, vtlxy, vtlxz, vtlyy, vtlyz, vtlzz;
   MAYBE_UNUSED real gxi, gyi, gzi, txi, tyi, tzi;
   MAYBE_UNUSED __shared__ real gxk[BLOCK_DIM], gyk[BLOCK_DIM], gzk[BLOCK_DIM],
      txk[BLOCK_DIM], tyk[BLOCK_DIM], tzk[BLOCK_DIM];


   for (int iw = iwarp; iw < niak; iw += nwarp) {
      if CONSTEXPR (do_a)
         ctl = 0;
      if CONSTEXPR (do_e)
         etl = 0;
      if CONSTEXPR (do_v) {
         vtlxx = 0;
         vtlxy = 0;
         vtlxz = 0;
         vtlyy = 0;
         vtlyz = 0;
         vtlzz = 0;
      }
      if CONSTEXPR (do_g) {
         gxi = 0;
         gyi = 0;
         gzi = 0;
         txi = 0;
         tyi = 0;
         tzi = 0;
         gxk[threadIdx.x] = 0;
         gyk[threadIdx.x] = 0;
         gzk[threadIdx.x] = 0;
         txk[threadIdx.x] = 0;
         tyk[threadIdx.x] = 0;
         tzk[threadIdx.x] = 0;
      }


      int atomi;
      atomi = min(iak[iw] * WARP_SIZE + ilane, n - 1);
      real xi = sorted[atomi].x;
      real yi = sorted[atomi].y;
      real zi = sorted[atomi].z;
      int i = sorted[atomi].unsorted;
      real ci = rpole[i][mpl_pme_0];
      real dix = rpole[i][mpl_pme_x];
      real diy = rpole[i][mpl_pme_y];
      real diz = rpole[i][mpl_pme_z];
      real qixx = rpole[i][mpl_pme_xx];
      real qixy = rpole[i][mpl_pme_xy];
      real qixz = rpole[i][mpl_pme_xz];
      real qiyy = rpole[i][mpl_pme_yy];
      real qiyz = rpole[i][mpl_pme_yz];
      real qizz = rpole[i][mpl_pme_zz];


      int shatomk;
      shatomk = lst[iw * WARP_SIZE + ilane];
      real shx = sorted[shatomk].x;
      real shy = sorted[shatomk].y;
      real shz = sorted[shatomk].z;
      int shk = sorted[shatomk].unsorted;
      real shck = rpole[shk][mpl_pme_0];
      real shdkx = rpole[shk][mpl_pme_x];
      real shdky = rpole[shk][mpl_pme_y];
      real shdkz = rpole[shk][mpl_pme_z];
      real shqkxx = rpole[shk][mpl_pme_xx];
      real shqkxy = rpole[shk][mpl_pme_xy];
      real shqkxz = rpole[shk][mpl_pme_xz];
      real shqkyy = rpole[shk][mpl_pme_yy];
      real shqkyz = rpole[shk][mpl_pme_yz];
      real shqkzz = rpole[shk][mpl_pme_zz];


      for (int j = 0; j < WARP_SIZE; ++j) {
         int srclane = (ilane + j) & (WARP_SIZE - 1);
         int atomk = __shfl_sync(ALL_LANES, shatomk, srclane);
         real xr = __shfl_sync(ALL_LANES, shx, srclane) - xi;
         real yr = __shfl_sync(ALL_LANES, shy, srclane) - yi;
         real zr = __shfl_sync(ALL_LANES, shz, srclane) - zi;
         int k = __shfl_sync(ALL_LANES, shk, srclane);
         real ck = __shfl_sync(ALL_LANES, shck, srclane);
         real dkx = __shfl_sync(ALL_LANES, shdkx, srclane);
         real dky = __shfl_sync(ALL_LANES, shdky, srclane);
         real dkz = __shfl_sync(ALL_LANES, shdkz, srclane);
         real qkxx = __shfl_sync(ALL_LANES, shqkxx, srclane);
         real qkxy = __shfl_sync(ALL_LANES, shqkxy, srclane);
         real qkxz = __shfl_sync(ALL_LANES, shqkxz, srclane);
         real qkyy = __shfl_sync(ALL_LANES, shqkyy, srclane);
         real qkyz = __shfl_sync(ALL_LANES, shqkyz, srclane);
         real qkzz = __shfl_sync(ALL_LANES, shqkzz, srclane);


         real e = 0;
         PairMPoleGrad pgrad;
         zero(pgrad);


         image(xr, yr, zr, box);
         real r2 = xr * xr + yr * yr + zr * zr;
         if (atomi < atomk && r2 <= off2) {
            if CONSTEXPR (ETYP == elec_t::ewald) {
               pair_mpole<USE, elec_t::ewald>(
                  r2, xr, yr, zr, 1,                                     //
                  ci, dix, diy, diz, qixx, qixy, qixz, qiyy, qiyz, qizz, //
                  ck, dkx, dky, dkz, qkxx, qkxy, qkxz, qkyy, qkyz, qkzz, //
                  f, aewald, e, pgrad);
            }
            if CONSTEXPR (ETYP == elec_t::coulomb) {
               pair_mpole<USE, elec_t::coulomb>(
                  r2, xr, yr, zr, 1,                                     //
                  ci, dix, diy, diz, qixx, qixy, qixz, qiyy, qiyz, qizz, //
                  ck, dkx, dky, dkz, qkxx, qkxy, qkxz, qkyy, qkyz, qkzz, //
                  f, 0, e, pgrad);
            }


            if CONSTEXPR (do_a)
               ctl += 1;
            if CONSTEXPR (do_e)
               etl += e;
            if CONSTEXPR (do_v) {
               vtlxx += -xr * pgrad.frcx;
               vtlxy += -0.5f * (yr * pgrad.frcx + xr * pgrad.frcy);
               vtlxz += -0.5f * (zr * pgrad.frcx + xr * pgrad.frcz);
               vtlyy += -yr * pgrad.frcy;
               vtlyz += -0.5f * (zr * pgrad.frcy + yr * pgrad.frcz);
               vtlzz += -zr * pgrad.frcz;
            }
         } // enf if (include)


         if CONSTEXPR (do_g) {
            gxi += pgrad.frcx;
            gyi += pgrad.frcy;
            gzi += pgrad.frcz;
            gxk[srclane + (threadIdx.x - ilane)] -= pgrad.frcx;
            gyk[srclane + (threadIdx.x - ilane)] -= pgrad.frcy;
            gzk[srclane + (threadIdx.x - ilane)] -= pgrad.frcz;


            txi += pgrad.ttmi[0];
            tyi += pgrad.ttmi[1];
            tzi += pgrad.ttmi[2];
            txk[srclane + (threadIdx.x - ilane)] += pgrad.ttmk[0];
            tyk[srclane + (threadIdx.x - ilane)] += pgrad.ttmk[1];
            tzk[srclane + (threadIdx.x - ilane)] += pgrad.ttmk[2];
         }
      } // end for (j)


      if CONSTEXPR (do_a)
         atomic_add(ctl, nem, offset);
      if CONSTEXPR (do_e)
         atomic_add(etl, em, offset);
      if CONSTEXPR (do_g) {
         atomic_add(gxi, gx, i);
         atomic_add(gyi, gy, i);
         atomic_add(gzi, gz, i);
         atomic_add(txi, trqx, i);
         atomic_add(tyi, trqy, i);
         atomic_add(tzi, trqz, i);
         atomic_add(gxk[threadIdx.x], gx, shk);
         atomic_add(gyk[threadIdx.x], gy, shk);
         atomic_add(gzk[threadIdx.x], gz, shk);
         atomic_add(txk[threadIdx.x], trqx, shk);
         atomic_add(tyk[threadIdx.x], trqy, shk);
         atomic_add(tzk[threadIdx.x], trqz, shk);
      }
      if CONSTEXPR (do_v)
         atomic_add(vtlxx, vtlxy, vtlxz, vtlyy, vtlyz, vtlzz, vir_em, offset);
   } // end for (iw)
}


template <int USE>
__global__
void empole_cu2(EMPOLE_ARGS, const real* restrict x, const real* restrict y,
                const real* restrict z, int nmexclude_,
                const int (*restrict mexclude_)[2],
                const real* restrict mexclude_scale_)
{
   constexpr int do_e = USE & calc::energy;
   constexpr int do_a = USE & calc::analyz;
   constexpr int do_g = USE & calc::grad;
   constexpr int do_v = USE & calc::virial;


   for (int ii = threadIdx.x + blockIdx.x * blockDim.x; ii < nmexclude_;
        ii += blockDim.x * gridDim.x) {
      int offset = ii & (bufsize - 1);


      int i = mexclude_[ii][0];
      int k = mexclude_[ii][1];
      real mscale = mexclude_scale_[ii];


      real xi = x[i];
      real yi = y[i];
      real zi = z[i];
      real ci = rpole[i][mpl_pme_0];
      real dix = rpole[i][mpl_pme_x];
      real diy = rpole[i][mpl_pme_y];
      real diz = rpole[i][mpl_pme_z];
      real qixx = rpole[i][mpl_pme_xx];
      real qixy = rpole[i][mpl_pme_xy];
      real qixz = rpole[i][mpl_pme_xz];
      real qiyy = rpole[i][mpl_pme_yy];
      real qiyz = rpole[i][mpl_pme_yz];
      real qizz = rpole[i][mpl_pme_zz];


      real xr = x[k] - xi;
      real yr = y[k] - yi;
      real zr = z[k] - zi;
      image(xr, yr, zr, box);
      real r2 = xr * xr + yr * yr + zr * zr;
      if (r2 <= off2) {
         real e;
         PairMPoleGrad pgrad;
         pair_mpole<USE, elec_t::coulomb>(
            r2, xr, yr, zr, mscale, ci, dix, diy, diz, qixx, qixy, qixz, qiyy,
            qiyz, qizz, rpole[k][mpl_pme_0], rpole[k][mpl_pme_x],
            rpole[k][mpl_pme_y], rpole[k][mpl_pme_z], rpole[k][mpl_pme_xx],
            rpole[k][mpl_pme_xy], rpole[k][mpl_pme_xz], rpole[k][mpl_pme_yy],
            rpole[k][mpl_pme_yz], rpole[k][mpl_pme_zz], f, 0, e, pgrad);


         if CONSTEXPR (do_a) {
            if (mscale == -1)
               atomic_add(-1, nem, offset);
         }
         if CONSTEXPR (do_e) {
            atomic_add(e, em, offset);
         }
         if CONSTEXPR (do_g) {
            atomic_add(pgrad.frcx, gx, i);
            atomic_add(pgrad.frcy, gy, i);
            atomic_add(pgrad.frcz, gz, i);
            atomic_add(-pgrad.frcx, gx, k);
            atomic_add(-pgrad.frcy, gy, k);
            atomic_add(-pgrad.frcz, gz, k);


            atomic_add(pgrad.ttmi[0], trqx, i);
            atomic_add(pgrad.ttmi[1], trqy, i);
            atomic_add(pgrad.ttmi[2], trqz, i);
            atomic_add(pgrad.ttmk[0], trqx, k);
            atomic_add(pgrad.ttmk[1], trqy, k);
            atomic_add(pgrad.ttmk[2], trqz, k);
         }
         if CONSTEXPR (do_v) {
            real vxx = -xr * pgrad.frcx;
            real vxy = -0.5f * (yr * pgrad.frcx + xr * pgrad.frcy);
            real vxz = -0.5f * (zr * pgrad.frcx + xr * pgrad.frcz);
            real vyy = -yr * pgrad.frcy;
            real vyz = -0.5f * (zr * pgrad.frcy + yr * pgrad.frcz);
            real vzz = -zr * pgrad.frcz;
            atomic_add(vxx, vxy, vxz, vyy, vyz, vzz, vir_em, offset);
         }
      } // end if (r2 <= off2)
   }
}


template <int USE, elec_t ETYP>
void empole_tmpl_cu()
{
   constexpr int do_e = USE & calc::energy;
   constexpr int do_a = USE & calc::analyz;


   const auto& st = *mspatial_unit;
   const real off = st.cutoff;
   const real off2 = off * off;
   auto bufsize = buffer_size();


   const real f = electric / dielec;
   real aewald = 0;
   if CONSTEXPR (ETYP == elec_t::ewald) {
      PMEUnit pu = epme_unit;
      aewald = pu->aewald;


      if (do_e && !do_a) {
         auto ker0e = empole_self_cu<calc::energy>;
         launch_kernel1(n, ker0e, //
                        bufsize, nem, em, rpole, n, f, aewald);
      } else if (do_e && do_a) {
         auto ker0a = empole_self_cu<calc::energy | calc::analyz>;
         launch_kernel1(n, ker0a, //
                        bufsize, nem, em, rpole, n, f, aewald);
      }
   }
   if (st.niak > 0) {
      auto ker1 = empole_cu1<USE, ETYP>;
      launch_kernel1(WARP_SIZE * st.niak, ker1, //
                     bufsize, nem, em, vir_em, gx, gy, gz, trqx, trqy, trqz,
                     box, off2, f, rpole, //
                     st.sorted, st.niak, st.iak, st.lst, n, aewald);
   }
   if (nmexclude_ > 0) {
      auto ker2 = empole_cu2<USE>;
      launch_kernel1(nmexclude_, ker2, //
                     bufsize, nem, em, vir_em, gx, gy, gz, trqx, trqy, trqz,
                     box, off2, f, rpole, //
                     x, y, z, nmexclude_, mexclude_, mexclude_scale_);
   }
}


template <int USE>
void empole_real_self_cu()
{
   empole_tmpl_cu<USE, elec_t::ewald>();
}
template void empole_real_self_cu<calc::v0>();
template void empole_real_self_cu<calc::v1>();
template void empole_real_self_cu<calc::v3>();
template void empole_real_self_cu<calc::v4>();
template void empole_real_self_cu<calc::v5>();
template void empole_real_self_cu<calc::v6>();


void empole_coulomb_cu(int vers)
{
   if (vers == calc::v0) {
      empole_tmpl_cu<calc::v0, elec_t::coulomb>();
   } else if (vers == calc::v1) {
      empole_tmpl_cu<calc::v1, elec_t::coulomb>();
   } else if (vers == calc::v3) {
      empole_tmpl_cu<calc::v3, elec_t::coulomb>();
   } else if (vers == calc::v4) {
      empole_tmpl_cu<calc::v4, elec_t::coulomb>();
   } else if (vers == calc::v5) {
      empole_tmpl_cu<calc::v5, elec_t::coulomb>();
   } else if (vers == calc::v6) {
      empole_tmpl_cu<calc::v6, elec_t::coulomb>();
   }
}
TINKER_NAMESPACE_END
