#include "hip/hip_runtime.h"
#include "box.h"
#include "launch.h"
#include "md.h"
#include "nblist.h"
#include "seq_image.h"
#include "seq_spatial_box.h"
#include "spatial.h"
#include "syntax/cu/ffsn.h"
#include "thrust_cache.h"
#include <thrust/extrema.h>
#include <thrust/remove.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/transform_reduce.h>
#include <thrust/transform_scan.h>


TINKER_NAMESPACE_BEGIN
namespace pbc {
#pragma acc routine seq
__device__
inline real3 c0_f0_triclinic(real xr, real yr, real zr, real3 ra, real3 rb,
                             real3 rc)
{
   real3 f;
   f.x = zr * ra.z + yr * ra.y + xr * ra.x;
   f.y = zr * rb.z + yr * rb.y;
   f.z = zr * rc.z;
   return f;
}


#pragma acc routine seq
__device__
inline real3 c0_f0_monoclinic(real xr, real yr, real zr, real3 ra, real3 rb,
                              real3 rc)
{
   real3 f;
   f.x = zr * ra.z + xr * ra.x;
   f.y = yr * rb.y;
   f.z = zr * rc.z;
   return f;
}


#pragma acc routine seq
__device__
inline real3 c0_f0_orthogonal(real xr, real yr, real zr, real3 ra, real3 rb,
                              real3 rc)
{
   real3 f;
   f.x = xr * ra.x;
   f.y = yr * rb.y;
   f.z = zr * rc.z;
   return f;
}


#pragma acc routine seq
__device__
inline real3 f0_f1(real3 f)
{
   f.x -= REAL_FLOOR(0.5f + f.x);
   f.y -= REAL_FLOOR(0.5f + f.y);
   f.z -= REAL_FLOOR(0.5f + f.z);
   return f;
}


#pragma acc routine seq
__device__
inline real3 f1_c1_triclinic(real3 f, real3 l1, real3 l2, real3 l3)
{
   f.x = f.z * l1.z + f.y * l1.y + f.x * l1.x;
   f.y = f.z * l2.z + f.y * l2.y;
   f.z = f.z * l3.z;
   return f;
}

#pragma acc routine seq
__device__
inline real3 f1_c1_monoclinic(real3 f, real3 l1, real3 l2, real3 l3)
{
   f.x = f.z * l1.z + f.x * l1.x;
   f.y = f.y * l2.y;
   f.z = f.z * l3.z;
   return f;
}

#pragma acc routine seq
__device__
inline real3 f1_c1_orthogonal(real3 f, real3 l1, real3 l2, real3 l3)
{
   f.x = f.x * l1.x;
   f.y = f.y * l2.y;
   f.z = f.z * l3.z;
   return f;
}
}

__device__
static inline real3 frac_general(real xr, real yr, real zr, real3 ra, real3 rb,
                                 real3 rc)
{
   using namespace pbc;
   if (ra.z == 0) {
      return f0_f1(c0_f0_orthogonal(xr, yr, zr, ra, rb, rc));
   } else if (ra.y == 0) {
      return f0_f1(c0_f0_monoclinic(xr, yr, zr, ra, rb, rc));
   } else {
      return f0_f1(c0_f0_triclinic(xr, yr, zr, ra, rb, rc));
   }
}


__device__
static inline real3 frac_image_general(real3 f, real3 l1, real3 l2, real3 l3)
{
   using namespace pbc;
   if (l1.z == 0) {
      return f1_c1_orthogonal(f, l1, l2, l3);
   } else if (l1.y == 0) {
      return f1_c1_monoclinic(f, l1, l2, l3);
   } else {
      return f1_c1_triclinic(f, l1, l2, l3);
   }
}


/**
 * \def frac
 * \ingroup macro
 * Calculate the fractional coordinates of (`xr, yr, zr`). The range of the
 * fractional coordinate is `[-1/2, 1/2)`.
 */
#ifndef frac
#   define frac(xr, yr, zr) frac_general(xr, yr, zr, recipa, recipb, recipc)
#endif
/**
 * \def frac_image
 * \ingroup macro
 */
#ifndef frac_image
#   define frac_image(f) frac_image_general(f, lvec1, lvec2, lvec3)
#endif


struct POPC
{
   __device__
   int operator()(int flag)
   {
      return __popc(flag);
   }
};


struct Int32
{
   long4 lx, ly, lz, lw;


   __device__
   static bool is_long4_zero(const long4& l)
   {
      return l.x == 0 && l.y == 0 && l.z == 0 && l.w == 0;
   }


   __device__
   static bool is_zero(const Int32& i32)
   {
      return is_long4_zero(i32.lx) && is_long4_zero(i32.ly) &&
         is_long4_zero(i32.lz) && is_long4_zero(i32.lw);
   }
};


struct IntInt32Pair
{
   struct Int32IsZero
   {
      __device__
      bool operator()(const thrust::tuple<int, Int32>& t)
      {
         return Int32::is_zero(thrust::get<1>(t));
      }
   };
};


__device__
bool nearby_box0(int boxj, int px, int py, int pz, real3 lvec1, real3 lvec2,
                 real3 lvec3, real cutbuf2)
{
   int dimx = 1 << px;
   int dimy = 1 << py;
   int dimz = 1 << pz;
   int ix, iy, iz;
   box_to_ixyz(ix, iy, iz, px, py, pz, boxj);

   // (a, b): (-0.5, a+1/dim)
   // (c, d): (a+ix/dim, c+1/dim)
   // da = a+(ix+1)/dim - a = (ix+1)/dim
   // cb = a+ix/dim - a-1/dim = (ix-1)/dim
   // min(image(da), image(cb))
   real3 r = make_real3(0, 0, 0);
   if (2 <= ix && ix <= dimx - 2) {
      real da = ((real)ix + 1) / dimx;
      real cb = ((real)ix - 1) / dimx;
      da -= REAL_FLOOR(da + 0.5f);
      cb -= REAL_FLOOR(cb + 0.5f);
      r.x = REAL_MIN(REAL_ABS(da), REAL_ABS(cb));
   }
   if (2 <= iy && iy <= dimy - 2) {
      real da = ((real)iy + 1) / dimy;
      real cb = ((real)iy - 1) / dimy;
      da -= REAL_FLOOR(da + 0.5f);
      cb -= REAL_FLOOR(cb + 0.5f);
      r.y = REAL_MIN(REAL_ABS(da), REAL_ABS(cb));
   }
   if (2 <= iz && iz <= dimz - 2) {
      real da = ((real)iz + 1) / dimz;
      real cb = ((real)iz - 1) / dimz;
      da -= REAL_FLOOR(da + 0.5f);
      cb -= REAL_FLOOR(cb + 0.5f);
      r.z = REAL_MIN(REAL_ABS(da), REAL_ABS(cb));
   }
   r = frac_image(r);
   real r2 = r.x * r.x + r.y * r.y + r.z * r.z;
   return r2 <= cutbuf2;
}


__device__
inline int offset_box(int nx, int ny, int nz, int ix1, int iy1, int iz1,
                      int offset)
{
   int dimx = (1 << nx);
   int dimy = (1 << ny);
   int dimz = (1 << nz);
   int ix, iy, iz;
   box_to_ixyz(ix, iy, iz, nx, ny, nz, offset);
   ix = (ix + ix1) & (dimx - 1);
   iy = (iy + iy1) & (dimy - 1);
   iz = (iz + iz1) & (dimz - 1);
   int id = ixyz_to_box(nx, ny, nz, ix, iy, iz);
   return id;
}


extern "C" __global__
void spatial_bc(int n, int px, int py, int pz,
                Spatial::SortedAtom* restrict sorted, int* restrict boxnum,
                int* restrict nax, //
                const real* restrict x, const real* restrict y,
                const real* restrict z, TINKER_IMAGE_PARAMS, real cutbuf2,
                int ZERO_LBUF, real* restrict xold, real* restrict yold,
                real* restrict zold, //
                int nx, int* restrict nearby)
{
   for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n;
        i += blockDim.x * gridDim.x) {
      real xr = x[i];
      real yr = y[i];
      real zr = z[i];
      if (!ZERO_LBUF) {
         xold[i] = xr;
         yold[i] = yr;
         zold[i] = zr;
      }
      real3 f = frac(xr, yr, zr);
      sorted[i].x = xr;       // B.2
      sorted[i].y = yr;       // B.2
      sorted[i].z = zr;       // B.2
      sorted[i].unsorted = i; // B.2
      int id = frac_to_box(px, py, pz, f.x, f.y, f.z);
      boxnum[i] = id;         // B.3
      atomicAdd(&nax[id], 1); // B.4
   }


   for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < nx;
        i += blockDim.x * gridDim.x) {
      if (nearby_box0(i, px, py, pz, lvec1, lvec2, lvec3, cutbuf2))
         nearby[i] = i; // C.1 (close enough)
      else
         nearby[i] = -1; // C.1 (otherwise)
   }
}


extern "C" __global__
void spatial_e(int n, int nak, const int* restrict boxnum, int* xakf,
               const Spatial::SortedAtom* restrict sorted, TINKER_IMAGE_PARAMS)
{
   const int ithread = threadIdx.x + blockIdx.x * blockDim.x;
   const int iwarp = ithread / WARP_SIZE;
   const int nwarp = blockDim.x * gridDim.x / WARP_SIZE;
   const int ilane = threadIdx.x & (WARP_SIZE - 1);
   const int prevlane = (ilane + WARP_SIZE - 1) & (WARP_SIZE - 1); // E.2


   for (int iw = iwarp; iw < nak; iw += nwarp) {
      int atomi = iw * WARP_SIZE + ilane;
      int id1 = ((atomi < n) ? boxnum[atomi] : boxnum[n - 1]); // E.3
      int id0 = __shfl_sync(ALL_LANES, id1, prevlane);         // E.5
      int diff = (id0 == id1 ? 0 : 1);                         // E.1
      int flag = __ballot_sync(ALL_LANES, diff);               // E.6
      if (ilane == 0)
         xakf[iw] = (flag == 0 ? 1 : flag); // E.4
   }
}


extern "C" __global__
void spatial_ghi(Spatial* restrict sp, int n, TINKER_IMAGE_PARAMS, real cutbuf2)
{
   const int ithread = threadIdx.x + blockIdx.x * blockDim.x;
   const int iwarp = ithread / WARP_SIZE;
   const int nwarp = blockDim.x * gridDim.x / WARP_SIZE;
   const int ilane = threadIdx.x & (WARP_SIZE - 1);

   const int nak = sp->nak;
   const int px = sp->px;
   const int py = sp->py;
   const int pz = sp->pz;
   const int nxk = sp->nxk;
   const int near = sp->near;

   const auto* restrict boxnum = sp->boxnum;
   const auto* restrict xakf = sp->xakf;
   const auto* restrict xakf_scan = sp->xakf_scan;
   const auto* restrict nearby = sp->nearby;
   const auto* restrict begin = sp->ax_scan; // D.4
   const auto* restrict end = begin + 1;     // D.4

   auto* restrict iak = sp->iak;
   auto* restrict lst = sp->lst;
   auto* restrict naak = sp->naak;
   auto* restrict xkf = sp->xkf;

   for (int iw = iwarp; iw < nak; iw += nwarp) {
      int offset = xakf_scan[iw]; // F.5
      int flag = xakf[iw];        // E.7
      int nbox = __popc(flag);    // E.7

      auto* restrict iakbuf = iak + near * offset;             // G.4
      auto* restrict lstbuf = lst + near * offset * WARP_SIZE; // G.5
      auto* restrict ixkf = xkf + iw * nxk;                    // H.2
      const int atom_block_min = iw * WARP_SIZE;               // H.4
      for (int j = ilane; j < nbox * near; j += WARP_SIZE) {
         iakbuf[j] = iw;    // G.4
         int i0 = j / near; // the i-th least significant bit is i0 + 1
         int pos = ffsn(flag, i0 + 1) - 1;        // E.8
         int ibox = boxnum[iw * WARP_SIZE + pos]; // E.8
         int ix1, iy1, iz1;
         box_to_ixyz(ix1, iy1, iz1, px, py, pz, ibox);
         int j0 = nearby[j - i0 * near];
         int jbox = offset_box(px, py, pz, ix1, iy1, iz1, j0);
         // the (jbox%32)-th bit of the (jbox/32) flag will be set to 1
         int ii = jbox / WARP_SIZE;
         int jj = jbox & (WARP_SIZE - 1);
         int oldflag = atomicOr(&ixkf[ii], 1 << jj); // H.3
         // the atomicOr() will return the old value;
         // code in the following if body will only run
         // when the bit(ii,jj) gets set for the first time
         if ((oldflag & (1 << jj)) == 0) {
            // copy atoms in jbox to lstbuf
            int begin_i = begin[jbox];
            begin_i = max(atom_block_min + 1, begin_i);        // H.4
            int len = end[jbox] - begin_i;                     // H.5
            int start_pos = atomicAdd(&naak[iw], max(0, len)); // H.6
            // atomicAdd() will return the old value;
            // skip the loop if len is less than 1
            for (int kk = 0; kk < len; ++kk) {
               lstbuf[start_pos + kk] = begin_i + kk; // H.4
            }
         }
      }
   }


   const auto* restrict sorted = sp->sorted;
   for (int iw = iwarp; iw < nak; iw += nwarp) {
      int offset = xakf_scan[iw];
      const auto* restrict iakbuf = iak + near * offset;
      auto* restrict lstbuf = lst + near * offset * WARP_SIZE;
      int naak_coarse = naak[iw]; // I.1


      int start_pos = 0;
      int atomi;
      atomi = min(iakbuf[0] * WARP_SIZE + ilane, n - 1); // I.4
      real xi = sorted[atomi].x;
      real yi = sorted[atomi].y;
      real zi = sorted[atomi].z;
      int shatomk;
      real shx, shy, shz;
      int idx_max = (naak_coarse + WARP_SIZE - 1) / WARP_SIZE;
      idx_max *= WARP_SIZE; // I.2a
      for (int idx = ilane; idx < idx_max; idx += WARP_SIZE) {
         shatomk = lstbuf[idx]; // I.2b
         shx = sorted[shatomk].x;
         shy = sorted[shatomk].y;
         shz = sorted[shatomk].z;
         lstbuf[idx] = 0; // I.3


         int jflag = 0;
         for (int j = 0; j < WARP_SIZE; ++j) {
            int srclane = j;
            int atomk = __shfl_sync(ALL_LANES, shatomk, srclane);
            real xr = xi - __shfl_sync(ALL_LANES, shx, srclane);
            real yr = yi - __shfl_sync(ALL_LANES, shy, srclane);
            real zr = zi - __shfl_sync(ALL_LANES, shz, srclane);
            real rik2 = imagen2(xr, yr, zr);
            int ilane_incl_j =
               (atomi < atomk && rik2 <= cutbuf2) ? 1 : 0; // I.5
            int incl_j = __ballot_sync(ALL_LANES, ilane_incl_j);
            if (incl_j)
               jflag |= (1 << j); // I.5
         }


         int njbit = __popc(jflag);
         int jth = ffsn(jflag, ilane + 1) - 1;
         int atomnb = __shfl_sync(ALL_LANES, shatomk, jth); // I.6a
         if (ilane < njbit)
            lstbuf[start_pos + ilane] = atomnb; // I.6b
         start_pos += njbit;
      }
   }
}


extern "C" __global__
void spatial_update_sorted(int n, Spatial::SortedAtom* restrict sorted,
                           const real* restrict x, const real* restrict y,
                           const real* restrict z, TINKER_IMAGE_PARAMS)
{
   for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n;
        i += blockDim.x * gridDim.x) {
      int ia = sorted[i].unsorted;
      real xr = x[ia];
      real yr = y[ia];
      real zr = z[ia];
      image(xr, yr, zr);
      sorted[i].x = xr;
      sorted[i].y = yr;
      sorted[i].z = zr;
   }
}
TINKER_NAMESPACE_END


TINKER_NAMESPACE_BEGIN
void spatial_data_update_sorted(SpatialUnit u)
{
   auto& st = *u;
   launch_k1s(nonblk, n, spatial_update_sorted, n, st.sorted, st.x, st.y, st.z,
              TINKER_IMAGE_ARGS);
}


void spatial_data_init_cu(SpatialUnit u)
{
   assert(u->rebuild == 1);
   u->rebuild = 0;
   const real cutbuf = u->cutoff + u->buffer;
   const real cutbuf2 = cutbuf * cutbuf;
   const real lbuf = u->buffer;
   const int& nak = u->nak;
   const int padded = nak * Spatial::BLOCK;
   int& px = u->px;
   int& py = u->py;
   int& pz = u->pz;
   int& nx = u->nx;
   int& nxk = u->nxk;
   int& near = u->near;
   int& xak_sum = u->xak_sum;
   int& xak_sum_cap = u->xak_sum_cap;
   int& niak = u->niak;


   auto*& sorted = u->sorted;
   auto*& boxnum = u->boxnum;
   auto*& naak = u->naak;
   auto*& xakf = u->xakf;
   auto*& xakf_scan = u->xakf_scan;
   auto*& nearby = u->nearby;
   auto*& ax_scan = u->ax_scan;
   auto*& xkf = u->xkf;


   // auto policy = thrust::device;
   auto policy = thrust::cuda::par(thrust_cache).on(nonblk);


   // B.1 D.1
   device_array::zero(false, nx + 1, ax_scan);
   // B.2 B.3 B.4 C.1
   const auto* lx = u->x;
   const auto* ly = u->y;
   const auto* lz = u->z;
   int ZERO_LBUF = (lbuf <= 0 ? 1 : 0);
   launch_k1s(nonblk, n, spatial_bc,                      //
              n, px, py, pz, sorted, boxnum, ax_scan + 1, //
              lx, ly, lz, TINKER_IMAGE_ARGS, cutbuf2, ZERO_LBUF, u->xold,
              u->yold, u->zold, //
              nx, nearby);
   // find max(nax) and compare to Spatial::BLOCK
   // ax_scan[0] == 0 can never be the maximum
   int level = 1 + floor_log2(nak - 1);
   int mnax;
   const int* mnaxptr = thrust::max_element(policy, ax_scan, ax_scan + 1 + nx);
   device_array::copyout(1, &mnax, mnaxptr, false);
   while (mnax > Spatial::BLOCK) {
      device_array::deallocate(nearby, ax_scan, xkf);

      int scale = (mnax - 1) / Spatial::BLOCK;
      // mnax / mnax-1 / scale / 2^p / p
      // 33   / 32     / 1     / 2   / 1
      // 64   / 63     / 1     / 2   / 1
      // 65   / 64     / 2     / 4   / 2
      // 128  / 127    / 3     / 4   / 2
      // 129  / 128    / 4     / 8   / 3
      int p = 1 + floor_log2(scale);
      level += p;
      px = (level + 0) / 3;
      py = (level + 1) / 3;
      pz = (level + 2) / 3;
      nx = pow2(px + py + pz);
      nxk = (nx + Spatial::BLOCK - 1) / Spatial::BLOCK;

      device_array::allocate(nx, &nearby);
      device_array::allocate(nx + 1, &ax_scan);
      device_array::allocate(nak * nxk, &xkf);

      u.update_deviceptr(*u, false);

      device_array::zero(false, nx + 1, ax_scan);
      int ZERO_LBUF = (lbuf <= 0 ? 1 : 0);
      launch_k1s(nonblk, n, spatial_bc,                      //
                 n, px, py, pz, sorted, boxnum, ax_scan + 1, //
                 lx, ly, lz, TINKER_IMAGE_ARGS, cutbuf2, ZERO_LBUF, u->xold,
                 u->yold, u->zold, //
                 nx, nearby);
      mnaxptr = thrust::max_element(policy, ax_scan, ax_scan + 1 + nx);
      device_array::copyout(1, &mnax, mnaxptr, false);
   }
   // B.5
   thrust::stable_sort_by_key(policy, boxnum, boxnum + n, sorted);
   // C.2
   int* nearby_end = thrust::remove(policy, nearby, nearby + nx, -1);
   // C.3
   near = nearby_end - nearby;
   // D.2
   int* nax = ax_scan + 1;
   // D.3
   thrust::inclusive_scan(policy, nax, nax + nx, nax);


   // E
   launch_k1s(nonblk, padded, spatial_e, n, nak, boxnum, xakf, sorted,
              TINKER_IMAGE_ARGS);
   // F.1
   xak_sum = thrust::transform_reduce(policy, xakf, xakf + nak, POPC(), 0,
                                      thrust::plus<int>());
   // F.2
   thrust::transform_exclusive_scan(policy, xakf, xakf + nak, xakf_scan, POPC(),
                                    0, thrust::plus<int>());
   if (xak_sum > xak_sum_cap) {
      device_array::deallocate(u->lst, u->iak);
      xak_sum_cap = xak_sum;
      size_t iak_size = near * xak_sum;            // F.3
      size_t lst_size = iak_size * Spatial::BLOCK; // F.4
      device_array::allocate(lst_size, &u->lst);
      device_array::allocate(iak_size, &u->iak);
   }
   // must update the device pointer to apply the changes in xak_sum
   u.update_deviceptr(*u, false);


   device_array::zero(false, near * xak_sum * Spatial::BLOCK,
                      u->lst);                // G.6
   device_array::zero(false, nak, naak);      // H.1
   device_array::zero(false, nak * nxk, xkf); // H.1
   launch_k1s(nonblk, padded, spatial_ghi, u.deviceptr(), n, TINKER_IMAGE_ARGS,
              cutbuf2);


   Int32* lst32 = (Int32*)u->lst;
   auto tup_begin =
      thrust::make_zip_iterator(thrust::make_tuple(u->iak, lst32));
   auto tup_end = thrust::make_zip_iterator(
      thrust::make_tuple(u->iak + near * xak_sum, lst32 + near * xak_sum));
   auto end2 = thrust::remove_if(policy, tup_begin, tup_end,
                                 IntInt32Pair::Int32IsZero());  // G.7
   u->niak = thrust::get<1>(end2.get_iterator_tuple()) - lst32; // G.7
   assert((thrust::get<0>(end2.get_iterator_tuple()) - u->iak) == u->niak);
   u.update_deviceptr(*u, false);
}
TINKER_NAMESPACE_END
