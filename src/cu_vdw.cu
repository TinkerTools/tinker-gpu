#include "hip/hip_runtime.h"
#include "add.cuh"
#include "e_vdw.h"
#include "launch.cuh"
#include "md.h"
#include "seq_image.h"
#include "seq_pair_hal.h"
#include "seq_switch.h"
#include "spatial.h"


/**
 * Overheads:
 *    - Different vcouple methods.
 *    - PBC type in image().
 *    - Random access to the "i" parameters and gradients.
 *    - (If not hard-coded) ghal, dhal, scexp, scalpha.
 */


TINKER_NAMESPACE_BEGIN
#define HAL_ARGS                                                               \
   size_t bufsize, count_buffer restrict nev, energy_buffer restrict ev,       \
      virial_buffer restrict vir_ev, real *restrict gxred,                     \
      real *restrict gyred, real *restrict gzred, TINKER_IMAGE_PARAMS,         \
      int njvdw, const int *restrict jvdw, const real *restrict radmin,        \
      const real *restrict epsilon, const real *vlam, evdw_t vcouple,          \
      real cut, real off
#if 1
#   define GHAL (real)0.12
#   define DHAL (real)0.07
#   define SCEXP 5
#   define SCALPHA (real)0.7
#elif 0
#   define GHAL ghal
#   define DHAL dhal
#   define SCEXP scexp
#   define SCALPHA scalpha
#endif
template <int USE>
__launch_bounds__(BLOCK_DIM) __global__
void evdw_hal_cu1(HAL_ARGS, int n, const Spatial::SortedAtom* restrict sorted,
                  int niak, const int* restrict iak, const int* restrict lst)
{
   constexpr int do_e = USE & calc::energy;
   constexpr int do_a = USE & calc::analyz;
   constexpr int do_g = USE & calc::grad;
   constexpr int do_v = USE & calc::virial;


   const int ithread = threadIdx.x + blockIdx.x * blockDim.x;
   const int iwarp = ithread / WARP_SIZE;
   const int nwarp = blockDim.x * gridDim.x / WARP_SIZE;
   const int ilane = threadIdx.x & (WARP_SIZE - 1);
   const int offset = ithread & (bufsize - 1);


   // thread local variables
   MAYBE_UNUSED int ctl;
   MAYBE_UNUSED real etl;
   MAYBE_UNUSED real gxi, gyi, gzi, gxk, gyk, gzk;
   MAYBE_UNUSED real vtlxx, vtlyx, vtlzx, vtlyy, vtlzy, vtlzz;


   const real cut2 = cut * cut;
   const real off2 = off * off;
   for (int iw = iwarp; iw < niak; iw += nwarp) {
      if CONSTEXPR (do_a)
         ctl = 0;
      if CONSTEXPR (do_e)
         etl = 0;
      if CONSTEXPR (do_g) {
         gxi = 0;
         gyi = 0;
         gzi = 0;
         gxk = 0;
         gyk = 0;
         gzk = 0;
      }
      if CONSTEXPR (do_v) {
         vtlxx = 0;
         vtlyx = 0;
         vtlzx = 0;
         vtlyy = 0;
         vtlzy = 0;
         vtlzz = 0;
      }


      int atomi = min(iak[iw] * WARP_SIZE + ilane, n - 1);
      real xi = sorted[atomi].x;
      real yi = sorted[atomi].y;
      real zi = sorted[atomi].z;
      int i = sorted[atomi].unsorted;
      int it = jvdw[i];
      real lam1 = vlam[i];


      int shatomk = lst[iw * WARP_SIZE + ilane];
      real shx = sorted[shatomk].x;
      real shy = sorted[shatomk].y;
      real shz = sorted[shatomk].z;
      int shk = sorted[shatomk].unsorted;
      int shkt = jvdw[shk];
      real shlam = vlam[shk];


      for (int j = 0; j < WARP_SIZE; ++j) {
         int srclane = (ilane + j) & (WARP_SIZE - 1);
         int atomk = __shfl_sync(ALL_LANES, shatomk, srclane);
         real xr = xi - __shfl_sync(ALL_LANES, shx, srclane);
         real yr = yi - __shfl_sync(ALL_LANES, shy, srclane);
         real zr = zi - __shfl_sync(ALL_LANES, shz, srclane);
         int kt = __shfl_sync(ALL_LANES, shkt, srclane);
         real vlambda = __shfl_sync(ALL_LANES, shlam, srclane);


         MAYBE_UNUSED real dedx = 0, dedy = 0, dedz = 0;
         real rik2 = image2(xr, yr, zr);


         if (atomi < atomk && rik2 <= off2) {
            real rik = REAL_SQRT(rik2);
            real rv = radmin[it * njvdw + kt];
            real eps = epsilon[it * njvdw + kt];
            if (vcouple == evdw_t::decouple) {
               vlambda = (lam1 == vlambda ? 1 : REAL_MIN(lam1, vlambda));
            } else if (vcouple == evdw_t::annihilate) {
               vlambda = REAL_MIN(lam1, vlambda);
            }


            MAYBE_UNUSED real e, de;
            pair_hal<do_g>(rik, rv, eps, 1, vlambda, GHAL, DHAL, SCEXP, SCALPHA,
                           e, de);
            if (rik2 > cut2) {
               real taper, dtaper;
               switch_taper5<do_g>(rik, cut, off, taper, dtaper);
               if CONSTEXPR (do_g)
                  de = e * dtaper + de * taper;
               if CONSTEXPR (do_e)
                  e = e * taper;
            }


            if CONSTEXPR (do_a)
               ctl += 1;
            if CONSTEXPR (do_e)
               etl += e;
            if CONSTEXPR (do_g) {
               de *= REAL_RECIP(rik);
               dedx = de * xr;
               dedy = de * yr;
               dedz = de * zr;
               if CONSTEXPR (do_v) {
                  vtlxx += xr * dedx;
                  vtlyx += yr * dedx;
                  vtlzx += zr * dedx;
                  vtlyy += yr * dedy;
                  vtlzy += zr * dedy;
                  vtlzz += zr * dedz;
               }
            }
         } // end if (include)


         if CONSTEXPR (do_g) {
            int dstlane = (ilane + WARP_SIZE - j) & (WARP_SIZE - 1);
            gxi += dedx;
            gyi += dedy;
            gzi += dedz;
            gxk -= __shfl_sync(ALL_LANES, dedx, dstlane);
            gyk -= __shfl_sync(ALL_LANES, dedy, dstlane);
            gzk -= __shfl_sync(ALL_LANES, dedz, dstlane);
         }
      }


      if CONSTEXPR (do_a)
         atomic_add(ctl, nev, offset);
      if CONSTEXPR (do_e)
         atomic_add(etl, ev, offset);
      if CONSTEXPR (do_g) {
         atomic_add(gxi, gxred, i);
         atomic_add(gyi, gyred, i);
         atomic_add(gzi, gzred, i);
         atomic_add(gxk, gxred, shk);
         atomic_add(gyk, gyred, shk);
         atomic_add(gzk, gzred, shk);
      }
      if CONSTEXPR (do_v)
         atomic_add(vtlxx, vtlyx, vtlzx, vtlyy, vtlzy, vtlzz, vir_ev, offset);
   } // end for (iw)
}


template <int USE>
__global__
void evdw_hal_cu2(HAL_ARGS, const real* restrict xred,
                  const real* restrict yred, const real* restrict zred,
                  int nvexclude_, int (*restrict vexclude_)[2],
                  real* restrict vexclude_scale_)
{
   constexpr int do_e = USE & calc::energy;
   constexpr int do_a = USE & calc::analyz;
   constexpr int do_g = USE & calc::grad;
   constexpr int do_v = USE & calc::virial;


   const real cut2 = cut * cut;
   const real off2 = off * off;
   for (int ii = threadIdx.x + blockIdx.x * blockDim.x; ii < nvexclude_;
        ii += blockDim.x * gridDim.x) {
      int offset = ii & (bufsize - 1);


      int i = vexclude_[ii][0];
      int k = vexclude_[ii][1];
      real vscale = vexclude_scale_[ii];


      int it = jvdw[i];
      real xi = xred[i];
      real yi = yred[i];
      real zi = zred[i];
      real lam1 = vlam[i];


      int kt = jvdw[k];
      real xr = xi - xred[k];
      real yr = yi - yred[k];
      real zr = zi - zred[k];
      real vlambda = vlam[k];


      real rik2 = image2(xr, yr, zr);
      if (rik2 <= off2) {
         real rik = REAL_SQRT(rik2);
         real rv = radmin[it * njvdw + kt];
         real eps = epsilon[it * njvdw + kt];
         if (vcouple == evdw_t::decouple) {
            vlambda = (lam1 == vlambda ? 1 : REAL_MIN(lam1, vlambda));
         } else if (vcouple == evdw_t::annihilate) {
            vlambda = REAL_MIN(lam1, vlambda);
         }


         MAYBE_UNUSED real e, de;
         pair_hal<do_g>(rik, rv, eps, vscale, vlambda, GHAL, DHAL, SCEXP,
                        SCALPHA, e, de);
         if (rik2 > cut2) {
            real taper, dtaper;
            switch_taper5<do_g>(rik, cut, off, taper, dtaper);
            if CONSTEXPR (do_g)
               de = e * dtaper + de * taper;
            if CONSTEXPR (do_e)
               e = e * taper;
         }


         if CONSTEXPR (do_a)
            if (vscale == -1)
               atomic_add(-1, nev, offset);
         if CONSTEXPR (do_e)
            atomic_add(e, ev, offset);
         if CONSTEXPR (do_g) {
            de *= REAL_RECIP(rik);
            real dedx = de * xr;
            real dedy = de * yr;
            real dedz = de * zr;
            atomic_add(dedx, gxred, i);
            atomic_add(dedy, gyred, i);
            atomic_add(dedz, gzred, i);
            atomic_add(-dedx, gxred, k);
            atomic_add(-dedy, gyred, k);
            atomic_add(-dedz, gzred, k);
            if CONSTEXPR (do_v) {
               real vxx = xr * dedx;
               real vyx = yr * dedx;
               real vzx = zr * dedx;
               real vyy = yr * dedy;
               real vzy = zr * dedy;
               real vzz = zr * dedz;
               atomic_add(vxx, vyx, vzx, vyy, vzy, vzz, vir_ev, offset);
            }
         }
      } // end if (include)
   }
}


template <int USE, evdw_t VDWTYP>
void evdw_cu()
{
   constexpr int do_e = USE & calc::energy;
   constexpr int do_a = USE & calc::analyz;
   constexpr int do_g = USE & calc::grad;
   constexpr int do_v = USE & calc::virial;
   static_assert(do_v ? do_g : true, "");
   static_assert(do_a ? do_e : true, "");

   const auto& st = *vspatial_unit;
   const real cut = switch_cut(switch_vdw);
   const real off = st.cutoff;
   const auto* sp = vspatial_unit.deviceptr();

   auto bufsize = buffer_size();

   if CONSTEXPR (do_g) {
      zero_gradient_async(n, gxred, gyred, gzred);
   }

   if CONSTEXPR (VDWTYP == evdw_t::hal) {
      if (st.niak > 0)
         launch_k1s(nonblk, WARP_SIZE * st.niak, evdw_hal_cu1<USE>, bufsize,
                    nev, ev, vir_ev, gxred, gyred, gzred, TINKER_IMAGE_ARGS,
                    njvdw, jvdw, radmin, epsilon, vlam, vcouple, cut, off, n,
                    st.sorted, st.niak, st.iak, st.lst);
      if (nvexclude_ > 0)
         launch_k1s(nonblk, nvexclude_, evdw_hal_cu2<USE>, bufsize, nev, ev,
                    vir_ev, gxred, gyred, gzred, TINKER_IMAGE_ARGS, njvdw, jvdw,
                    radmin, epsilon, vlam, vcouple, cut, off, xred, yred, zred,
                    nvexclude_, vexclude_, vexclude_scale_);
   }

   if CONSTEXPR (do_g) {
      evdw_resolve_gradient();
   }
}


void evdw_hal_cu(int vers)
{
   if (vers == calc::v0)
      evdw_cu<calc::v0, evdw_t::hal>();
   else if (vers == calc::v1)
      evdw_cu<calc::v1, evdw_t::hal>();
   else if (vers == calc::v3)
      evdw_cu<calc::v3, evdw_t::hal>();
   else if (vers == calc::v4)
      evdw_cu<calc::v4, evdw_t::hal>();
   else if (vers == calc::v5)
      evdw_cu<calc::v5, evdw_t::hal>();
   else if (vers == calc::v6)
      evdw_cu<calc::v6, evdw_t::hal>();
}
TINKER_NAMESPACE_END
